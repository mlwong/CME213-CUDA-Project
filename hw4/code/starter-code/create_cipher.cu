#include "hip/hip_runtime.h"
#include <algorithm>
#include <cctype>
#include <fstream>
#include <iostream>
#include <vector>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
// You may include other thrust headers if necessary.

#include <thrust/iterator/constant_iterator.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/sort.h>
#include <thrust/transform.h>

#include "test_macros.h"

// You will need to call these functors from
// thrust functions in the code do not create new ones

// returns true if the char is not a lowercase letter
struct isnot_lowercase_alpha : thrust::unary_function<unsigned char, bool> {
  // TODO
  
  __host__ __device__
  bool operator()(const unsigned char &character)
  {
    if (character < 'a' || character > 'z')
    {
      return true;
    }
    return false;
  }
  
};

// convert an uppercase letter into a lowercase one
// do not use the builtin C function or anything from boost, etc.
struct upper_to_lower : thrust::unary_function<unsigned char, unsigned char> {
  // TODO
  
  __host__ __device__
  unsigned char operator()(const unsigned char &character)
  {
    if (character >= 'A' && character <= 'Z')
    {
      return (character + 32);
    }
    else
    {
      return character;
    }
  }
  
};

// apply a shift with appropriate wrapping
struct apply_shift : thrust::binary_function<unsigned char, int,
                                             unsigned char> {
  // TODO
  
  unsigned int *shift_amounts;
  unsigned int period;
  
  __host__ __device__
  apply_shift(unsigned int *shift_amounts_input, unsigned int period_input):
                shift_amounts(shift_amounts_input),
                period(period_input)
  {}
  
  __host__ __device__
  unsigned char operator()(const unsigned char &character, const unsigned int &position)
  {
    unsigned char converted_character = character + shift_amounts[position%period];
    
    if (converted_character > 122)
    {
      converted_character -= 26;
    }
    
    return converted_character;    
  }
  
};

// Returns a vector with the top 5 letter frequencies in text.
std::vector<double> getLetterFrequencyCpu(
    const std::vector<unsigned char> &text) {
  std::vector<unsigned int> freq(256);
  for (unsigned int i = 0; i < text.size(); ++i)
    freq[tolower(text[i])]++;

  unsigned int sum_chars = 0;
  for (unsigned char c = 'a'; c <= 'z'; ++c)
    sum_chars += freq[c];

  std::vector<double> freq_alpha_lower;
  for (unsigned char c = 'a'; c <= 'z'; ++c) {
    if (freq[c] > 0)
      freq_alpha_lower.push_back(freq[c] / static_cast<double>(sum_chars));
  }

  std::sort(freq_alpha_lower.begin(), freq_alpha_lower.end(),
            std::greater<double>());
  freq_alpha_lower.resize(min(static_cast<int>(freq_alpha_lower.size()), 5));

  return freq_alpha_lower;
}

// Print the top 5 letter frequencies and them.
std::vector<double> getLetterFrequencyGpu(
    const thrust::device_vector<unsigned char> &text) {
  std::vector<double> freq_alpha_lower;
  // WARNING: make sure you handle the case of not all letters appearing
  // in the text.

  // TODO calculate letter frequency
  
  thrust::device_vector<unsigned char> characters = text;
  
  // assume the text has all 26 letters
  // find the number of occurence of each letter
  
  thrust::device_vector<unsigned char> histogram_keys(26);
  thrust::device_vector<int> histogram_values(26);
  
  thrust::sort(characters.begin(), characters.end());
  
  typedef thrust::device_vector<unsigned char>::iterator devIt_1;
  typedef thrust::device_vector<int>::iterator devIt_2;
  
  thrust::pair<devIt_1, devIt_2> endIterators =
    thrust::reduce_by_key(characters.begin(),
                          characters.end(),
                          thrust::make_constant_iterator(1),
                          histogram_keys.begin(),
                          histogram_values.begin());
  
  int num_letters = endIterators.first - histogram_keys.begin();
  
  // sort the letters according to descending order of number of
  // occurence of letters
  
  thrust::sort_by_key(histogram_values.begin(),
                      histogram_values.end(),
                      histogram_keys.begin(),
                      thrust::greater<int>());
  
  // output the top 5 letters along with their frequencies
  
  std::cout << "Top 5 Letter Frequencies" << std::endl;
  std::cout << "-------------" << std::endl;
    
  for (unsigned int i = 0; i < min(num_letters, 5); i++)
  {
    freq_alpha_lower.push_back(((double) histogram_values[i])/((double) text.size()));
    std::cout << histogram_keys[i]
              << " "
              << freq_alpha_lower[i]
              << std::endl;
  }
  
  return freq_alpha_lower;
}

int main(int argc, char **argv) {
  if (argc != 3) {
    std::cerr << "Didn't supply plain text and period!" << std::endl;
    return 1;
  }

  std::ifstream ifs(argv[1], std::ios::binary);
  if (!ifs.good()) {
    std::cerr << "Couldn't open text file!" << std::endl;
    return 1;
  }

  unsigned int period = atoi(argv[2]);

  if (period < 4) {
    std::cerr << "Period must be at least 4!" << std::endl;
    return 1;
  }

  // load the file into text
  std::vector<unsigned char> text;

  ifs.seekg(0, std::ios::end); // seek to end of file
  int length = ifs.tellg();    // get distance from beginning
  ifs.seekg(0, std::ios::beg); // move back to beginning

  text.resize(length);
  ifs.read((char *)&text[0], length);

  ifs.close();

  thrust::device_vector<unsigned char> text_clean;
  
  // TODO: sanitize input to contain only a-z lowercase (use the
  // isnot_lowercase_alpha functor), calculate the number of characters
  // in the cleaned text and put the result in text_clean, make sure to
  // resize text_clean to the correct size!
  int numElements = -1;
  
  thrust::device_vector<unsigned char> d_text = text;
  text_clean.resize(d_text.size());
  
  thrust::device_vector<unsigned char>::iterator endIterator =
    thrust::remove_copy_if(thrust::make_transform_iterator(d_text.begin(), upper_to_lower()),
                           thrust::make_transform_iterator(d_text.end(), upper_to_lower()),
                           text_clean.begin(),
                           isnot_lowercase_alpha());
  
  numElements = endIterator - text_clean.begin();
  
  text_clean.resize(numElements);
  
  std::cout << "\nBefore ciphering!" << std::endl << std::endl;
  std::vector<double> letterFreqGpu = getLetterFrequencyGpu(text_clean);
  std::vector<double> letterFreqCpu = getLetterFrequencyCpu(text);
  bool success = true;
  EXPECT_VECTOR_EQ_EPS(letterFreqCpu, letterFreqGpu, 1e-14, &success);
  PRINT_SUCCESS(success);

  thrust::device_vector<unsigned int> shifts(period);
  
  // TODO fill in shifts using thrust random number generation (make sure
  // not to allow 0-shifts, this would make for rather poor encryption).
  
  thrust::default_random_engine rng(123); // use seed of 123
  thrust::uniform_int_distribution<unsigned int> u_dist(1, 25);
  
  for (unsigned int i = 0; i < period; i++)
  {
    shifts[i] = u_dist(rng);
  }
  
  std::cout << "\nEncryption key: ";
  for (int i = 0; i < period; ++i)
    std::cout << static_cast<char>('a' + shifts[i]);
  std::cout << std::endl;

  thrust::device_vector<unsigned char> device_cipher_text(numElements);

  // TODO: Apply the shifts to text_clean and place the result in
  // device_cipher_text.
  
  apply_shift as(thrust::raw_pointer_cast(&shifts[0]), period);
  
  thrust::transform(text_clean.begin(),
                    text_clean.end(),
                    thrust::make_counting_iterator(0),
                    device_cipher_text.begin(),
                    as);
  
  thrust::host_vector<unsigned char> host_cipher_text = device_cipher_text;
  
  std::cout << "After ciphering!" << std::endl << std::endl;
  getLetterFrequencyGpu(device_cipher_text);

  std::ofstream ofs("cipher_text.txt", std::ios::binary);

  ofs.write((char *)&host_cipher_text[0], numElements);

  ofs.close();

  return 0;
}
