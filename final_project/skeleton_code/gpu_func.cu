#include "hip/hip_runtime.h"
#include "gpu_func.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <iostream>

__global__
void device_add_one (int* d_result, int t)
{
	*d_result = t + 1;
}


int useless_gpu_add_one (int t)
{
	int result;
	int *d_result;

	checkCudaErrors (hipMalloc((void **)&d_result, 1 * sizeof (int)));

	event_pair timer;
	start_timer (&timer);
	device_add_one<<<1,1>>>(d_result, t);
	check_launch ("device_add_one");
	double time = stop_timer (&timer);

	std::cout << "device_add_one took: " << time << " seconds" << std::endl;

	checkCudaErrors (hipMemcpy(&result, d_result, 1 * sizeof (int), hipMemcpyDeviceToHost));
	return result;
}