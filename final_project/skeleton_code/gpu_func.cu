#include "hip/hip_runtime.h"
#include "gpu_func.h"

#include <cmath>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <iostream>

// Block size used in transpose
#define BLOCK_SIZE_x_transpose 64
#define BLOCK_SIZE_y_transpose 8

// Block size used in algorithm 0 of GEMM
#define BLOCK_SIZE_0 256

// Block size used in algorithm 1 of GEMM
#define BLOCK_SIZE_x_1 32
#define BLOCK_SIZE_y_1 16

// Block size used in algorithm 2 of GEMM
#define BLOCK_SIZE_2 32

// Block size used in algorithm 3 of GEMM
#define BLOCK_SIZE_x_3 16
#define BLOCK_SIZE_y_3 4

// Block size used in sigmoid function
#define BLOCK_SIZE_x_SIGMOID 64
#define BLOCK_SIZE_y_SIGMOID 8

// Block size used in softmax function
#define BLOCK_SIZE_x_SOFTMAX 64
#define BLOCK_SIZE_y_SOFTMAX 8

// Block size used in reduction
#define BLOCK_SIZE_REDUCTION 32

// Block size used in elementwise multiplication
#define BLOCK_SIZE_x_elementwise_mult 64
#define BLOCK_SIZE_y_elementwise_mult 8

// Block size used in elementwise substraction
#define BLOCK_SIZE_x_elementwise_subtract 64
#define BLOCK_SIZE_y_elementwise_subtract 8

// Block size used in difference function
#define BLOCK_SIZE_x_DIFF 64
#define BLOCK_SIZE_y_DIFF 8

__global__
void device_add_one (int* d_result, int t)
{
	*d_result = t + 1;
}

int useless_gpu_add_one (int t)
{
	int result;
	int *d_result;

	checkCudaErrors (hipMalloc((void **)&d_result, 1 * sizeof (int)));

	event_pair timer;
	start_timer (&timer);
	device_add_one<<<1,1>>>(d_result, t);
	check_launch ("device_add_one");
	double time = stop_timer (&timer);

	std::cout << "device_add_one took: " << time << " seconds" << std::endl;

	checkCudaErrors (hipMemcpy(&result, d_result, 1 * sizeof (int), hipMemcpyDeviceToHost));
	return result;
}

// Kernel to compute the very naive GEMM algorithm
__global__
void device_GEMM_0 (const double alpha,
				    const double beta,
				    const double* const d_mat_A,
				    const double* const d_mat_B,
				    const double* const d_mat_C,
				    double* d_mat_D,
				    const int m,
				    const int n,
				    const int l,
					const bool transpose_A,
					const bool transpose_B)
{
	// Global thread index in the grid
	const int tid = threadIdx.x + blockDim.x*blockIdx.x;
	
	// Compute the column and row of the element that thread is
	// computing
	int col = tid%l;
	int row = tid/l;

	// If the thread is not inside the matrix D, return
	if (row >= m || col >= l)
	{
		return;
	}
	
	// Compute the index in matrix D
	int idx = col*m + row;
	
	// sum is used to store the element of op(A)*op(B)
	// that is computed by the thread
	double sum = 0.0;
	
	// Do the multiplication and summation
	if (transpose_A == false)
	{
		if (transpose_B == false)
		{
			for (int k = 0; k < n; k++)
			{
				int idx_A = k*m + row;
				int idx_B = col*n + k;
				
				sum += d_mat_A[idx_A]*d_mat_B[idx_B];
			}
		}
		else
		{
			for (int k = 0; k < n; k++)
			{
				int idx_A = k*m + row;
				int idx_B = k*l + col;
				
				sum += d_mat_A[idx_A]*d_mat_B[idx_B];
			}
		}
	}
	else
	{
		if (transpose_B == false)
		{
			for (int k = 0; k < n; k++)
			{
				int idx_A = row*n + k;
				int idx_B = col*n + k;
				
				sum += d_mat_A[idx_A]*d_mat_B[idx_B];
			}
		}
		else
		{
			for (int k = 0; k < n; k++)
			{
				int idx_A = row*n + k;
				int idx_B = k*l + col;
				
				sum += d_mat_A[idx_A]*d_mat_B[idx_B];
			}
		}
	}
	
	// Each thread writes one element of matrix D
	if (beta == 0.0)
	{
		d_mat_D[idx] = alpha*sum;
	}
	else
	{
		d_mat_D[idx] = alpha*sum + beta*d_mat_C[idx];
	}
}

/*
 * Algorithm 0 of general matrix-matrix multiplication (GEMM)
 * GEMM operation is expressed as D = alpha*op(A)*op(B) + beta*C
 * One thread is used to calculate one element in matrix D natively
 * 1D blocks are used
 * natively
 * 
 * Parameters:
 *  m:              Number of rows of op(A) / number of rows of C/D
 *  n:              Number of columns of op(A) / number of rows of op(B)
 *  l:              Number of columns of op(B) / number of columns of C/D
 *  transpose_A:    Whether A should be transposed
 *                  If transpose_A is false, op(A) = A
 *                  Otherwise, op(A) = A^T
 *  transpose_B:    Whether B should be transposed
 *                  If transpose_B is false, op(B) = B
 *                  Otherwise, op(B) = B^T
 */
void gpu_GEMM_0 (const double alpha,
                 const double beta,
                 const double* const mat_A,
                 const double* const mat_B,
                 const double* const mat_C,
                 double* mat_D,
			     const int m,
			     const int n,
			     const int l,
				 const bool transpose_A,
				 const bool transpose_B)
{
	double *d_mat_A;
	double *d_mat_B;
	double *d_mat_C;
	double *d_mat_D;
	
	// Allocate the device memory
	checkCudaErrors(hipMalloc(&d_mat_A, m*n*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_B, n*l*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_C, m*l*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_D, m*l*sizeof(double)));
	
	// Copy data from the host memory to the device memory
	checkCudaErrors(hipMemcpy(d_mat_A, mat_A, m*n*sizeof(double), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_mat_B, mat_B, n*l*sizeof(double), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_mat_C, mat_C, m*l*sizeof(double), hipMemcpyHostToDevice));

    const int num_blocks = (m*l + BLOCK_SIZE_0 - 1)/BLOCK_SIZE_0;

	// Launch the kernel
	device_GEMM_0 <<<num_blocks, BLOCK_SIZE_0>>> (alpha,
											      beta,
											      d_mat_A,
											      d_mat_B,
											      d_mat_C,
											      d_mat_D,
											      m,
											      n,
											      l,
											      transpose_A,
											      transpose_B);
	
	// Copy data from the device memory to the host memory
	checkCudaErrors(hipMemcpy(mat_D, d_mat_D, m*l*sizeof(double), hipMemcpyDeviceToHost));
	
	// Free the device memory
	hipFree(d_mat_A);
	hipFree(d_mat_B);
	hipFree(d_mat_C);
	hipFree(d_mat_D);
}

// Kernel to compute the first GEMM algorithm
__global__
void device_GEMM_1 (const double alpha,
				    const double beta,
				    const double* const d_mat_A,
				    const double* const d_mat_B,
				    const double* const d_mat_C,
				    double* d_mat_D,
				    const int m,
				    const int n,
				    const int l,
					const bool transpose_A,
					const bool transpose_B)
{
	// Global thread index in the grid
	const int tid_x = threadIdx.x + blockDim.x*blockIdx.x;
	const int tid_y = threadIdx.y + blockDim.y*blockIdx.y;
	
	// If the thread is not inside the matrix D, return
	if (tid_x >= m || tid_y >= l)
	{
		return;
	}
	
	// Compute the index in matrix D
	int idx = tid_y*m + tid_x;
	
	// sum is used to store the element of op(A)*op(B)
	// that is computed by the thread
	double sum = 0.0;
	
	// Do the multiplication and summation
	if (transpose_A == false)
	{
		if (transpose_B == false)
		{
			for (int k = 0; k < n; k++)
			{
				int idx_A = k*m + tid_x;
				int idx_B = tid_y*n + k;
				
				sum += d_mat_A[idx_A]*d_mat_B[idx_B];
			}
		}
		else
		{
			for (int k = 0; k < n; k++)
			{
				int idx_A = k*m + tid_x;
				int idx_B = k*l + tid_y;
				
				sum += d_mat_A[idx_A]*d_mat_B[idx_B];
			}
		}
	}
	else
	{
		if (transpose_B == false)
		{
			for (int k = 0; k < n; k++)
			{
				int idx_A = tid_x*n + k;
				int idx_B = tid_y*n + k;
				
				sum += d_mat_A[idx_A]*d_mat_B[idx_B];
			}
		}
		else
		{
			for (int k = 0; k < n; k++)
			{
				int idx_A = tid_x*n + k;
				int idx_B = k*l + tid_y;
				
				sum += d_mat_A[idx_A]*d_mat_B[idx_B];
			}
		}
	}
	
	// Each thread writes one element of matrix D
	if (beta == 0.0)
	{
		d_mat_D[idx] = alpha*sum;
	}
	else
	{
		d_mat_D[idx] = alpha*sum + beta*d_mat_C[idx];
	}
}

/*
 * Algorithm 1 of general matrix-matrix multiplication (GEMM)
 * GEMM operation is expressed as D = alpha*op(A)*op(B) + beta*C
 * One thread is used to calculate one element in matrix D
 * natively. 2D blocks are used
 * 
 * Parameters:
 *  m:              Number of rows of op(A) / number of rows of C/D
 *  n:              Number of columns of op(A) / number of rows of op(B)
 *  l:              Number of columns of op(B) / number of columns of C/D
 *  transpose_A:    Whether A should be transposed
 *                  If transpose_A is false, op(A) = A
 *                  Otherwise, op(A) = A^T
 *  transpose_B:    Whether B should be transposed
 *                  If transpose_B is false, op(B) = B
 *                  Otherwise, op(B) = B^T
 */
void gpu_GEMM_1 (const double alpha,
                 const double beta,
                 const double* const mat_A,
                 const double* const mat_B,
                 const double* const mat_C,
                 double* mat_D,
			     const int m,
			     const int n,
			     const int l,
				 const bool transpose_A,
				 const bool transpose_B)
{
	double *d_mat_A;
	double *d_mat_B;
	double *d_mat_C;
	double *d_mat_D;
	
	// Allocate the device memory
	checkCudaErrors(hipMalloc(&d_mat_A, m*n*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_B, n*l*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_C, m*l*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_D, m*l*sizeof(double)));
	
	// Copy data from the host memory to the device memory
	checkCudaErrors(hipMemcpy(d_mat_A, mat_A, m*n*sizeof(double), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_mat_B, mat_B, n*l*sizeof(double), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_mat_C, mat_C, m*l*sizeof(double), hipMemcpyHostToDevice));
	
	dim3 n_threads(0, 0);
	dim3 n_blocks(0, 0);
	
	// Compute the block dimension
	n_threads.x = BLOCK_SIZE_x_1;
	n_threads.y = BLOCK_SIZE_y_1;
	
	// Compute the grid size
	n_blocks.x = (m + n_threads.x - 1)/n_threads.x;
	n_blocks.y = (l + n_threads.y - 1)/n_threads.y;
	
	// Launch the kernel
	device_GEMM_1 <<<n_blocks, n_threads>>> (alpha,
											 beta,
											 d_mat_A,
											 d_mat_B,
											 d_mat_C,
											 d_mat_D,
											 m,
											 n,
											 l,
											 transpose_A,
											 transpose_B);
	
	// Copy data from the device memory to the host memory
	checkCudaErrors(hipMemcpy(mat_D, d_mat_D, m*l*sizeof(double), hipMemcpyDeviceToHost));
	
	// Free the device memory
	hipFree(d_mat_A);
	hipFree(d_mat_B);
	hipFree(d_mat_C);
	hipFree(d_mat_D);
}

// Kernel to compute the second GEMM algorithm
template <int block_size>
__global__
void device_GEMM_2 (const double alpha,
				    const double beta,
				    const double* const d_mat_A,
				    const double* const d_mat_B,
				    const double* const d_mat_C,
				    double* d_mat_D,
				    const int m,
				    const int n,
				    const int l,
					const bool transpose_A,
					const bool transpose_B)
{
	// Global thread index in the grid
	const int g_tid_x = threadIdx.x + blockDim.x*blockIdx.x;
	const int g_tid_y = threadIdx.y + blockDim.y*blockIdx.y;
	
	// Local thread index
	const int tid_x = threadIdx.x;
	const int tid_y = threadIdx.y;
	
	// Block index
	const int bid_x = blockIdx.x;
	const int bid_y = blockIdx.y;
	
	// Declaration of the shared memory used to store the
	// sub-matrix of A/A^T
	__shared__ double mat_A_shared[block_size+1][block_size+1];
	
	// Declaration of the shared memory used to store the
	// sub-matrix of B/B^T
	__shared__ double mat_B_shared[block_size+1][block_size+1];
	
	if (transpose_A == false)
	{
		if (transpose_B == false)
		{
			// Index of the first sub-matrix of A processed by the block
			int mat_A_begin = block_size*bid_x;
			
			// Step size used to iterate through the sub-matrices of A
			int mat_A_step  = block_size*m;
			
			// Index of the first sub-matrix of B processed by the block
			int mat_B_begin = n*block_size*bid_y;
			
			// Index of the last sub-matrix of B processed by the blcok
			int mat_B_end   = mat_B_begin + n - 1;
			
			// Step size used to iterate through the sub-matrices of B
			int mat_B_step  = block_size;
			
			// sum is used to store the element of the block sub-matrix
			// that is computed by the thread
			double sum = 0.0;
			
			// Counter to record the current positions of column position
			// of sub-matrix in matrix A and row position of sub-matrix
			// in matrix B
			int idx_A_col = 0;
			int idx_B_row = 0; 
			
			// Loop over all the sub-matrices of A and B
			// required to compute the block sub-matrix
			for (int idx_A = mat_A_begin, idx_B = mat_B_begin;
				 idx_B <= mat_B_end;
				 idx_A += mat_A_step, idx_B += mat_B_step)
			{
				// Load the matrices from device memory to shared memory;
				// Each thread loads one element of each sub-matrix
				
				if (g_tid_x < m && tid_y + idx_A_col < n)
					mat_A_shared[tid_x][tid_y] = d_mat_A[idx_A + m*tid_y + tid_x];
				if (tid_x + idx_B_row < n && g_tid_y < l)
					mat_B_shared[tid_x][tid_y] = d_mat_B[idx_B + n*tid_y + tid_x];
				
				// Synchronize to make sure the matrices are loaded
				__syncthreads();
				
				if (g_tid_x < m && g_tid_y < l)
				{
					int k_bound = min(block_size, n - idx_A_col);
					
					for (int k = 0; k < k_bound; k++)
					{
						sum += mat_A_shared[tid_x][k]*mat_B_shared[k][tid_y];
					}
				}
				
				idx_A_col += block_size;
				idx_B_row += block_size;
				
				// Synchronize to make sure that the preceding computation
				// is done before loading two new sub-matrices of A and B
				// in the next iteration
				__syncthreads();
			}
			
			// Write the block sub-matrix to device memory
			// each thread writes one element
			if (g_tid_x < m && g_tid_y < l)
			{
				int idx_D = m*block_size*bid_y + block_size*bid_x + tid_x + m*tid_y;
				
				if (beta == 0.0)
				{
					d_mat_D[idx_D] = alpha*sum;
				}
				else
				{
					d_mat_D[idx_D] = alpha*sum + beta*d_mat_C[idx_D];
				}
			}
		}
		else
		{
			// Index of the first sub-matrix of A processed by the block
			int mat_A_begin = block_size*bid_x;
			
			// Step size used to iterate through the sub-matrices of A
			int mat_A_step  = block_size*m;
			
			// Index of the first sub-matrix of B^T processed by the block
			int mat_B_t_begin = block_size*bid_y;
			
			// Index of the last sub-matrix of B^T processed by the blcok
			int mat_B_t_end   = mat_B_t_begin + (n - 1)*l;
			
			// Step size used to iterate through the sub-matrices of B^T
			int mat_B_t_step  = block_size*l;
			
			// sum is used to store the element of the block sub-matrix
			// that is computed by the thread
			double sum = 0.0;
			
			// Counter to record the current positions of column position
			// of sub-matrix in matrix A and row position of sub-matrix
			// in matrix B^T
			int idx_A_col = 0;
			int idx_B_t_row = 0; 
			
			// Loop over all the sub-matrices of A and B^T
			// required to compute the block sub-matrix
			for (int idx_A = mat_A_begin, idx_B_t = mat_B_t_begin;
				 idx_B_t <= mat_B_t_end;
				 idx_A += mat_A_step, idx_B_t += mat_B_t_step)
			{				
				// Load the matrices from device memory to shared memory;
				// Each thread loads one element of each sub-matrix
				
				if (g_tid_x < m && tid_y + idx_A_col < n)
					mat_A_shared[tid_x][tid_y] = d_mat_A[idx_A + m*tid_y + tid_x];
				if (tid_x + idx_B_t_row < n && g_tid_y < l)
					mat_B_shared[tid_x][tid_y] = d_mat_B[idx_B_t + l*tid_x + tid_y];
				
				// Synchronize to make sure the matrices are loaded
				__syncthreads();
				
				if (g_tid_x < m && g_tid_y < l)
				{
					int k_bound = min(block_size, n - idx_A_col);
					
					for (int k = 0; k < k_bound; k++)
					{
						sum += mat_A_shared[tid_x][k]*mat_B_shared[k][tid_y];
					}
				}
				
				idx_A_col += block_size;
				idx_B_t_row += block_size;
				
				// Synchronize to make sure that the preceding computation
				// is done before loading two new sub-matrices of A and B^T
				// in the next iteration
				__syncthreads();
			}
			
			// Write the block sub-matrix to device memory
			// each thread writes one element
			if (g_tid_x < m && g_tid_y < l)
			{
				int idx_D = m*block_size*bid_y + block_size*bid_x + tid_x + m*tid_y;
				
				if (beta == 0.0)
				{
					d_mat_D[idx_D] = alpha*sum;
				}
				else
				{
					d_mat_D[idx_D] = alpha*sum + beta*d_mat_C[idx_D];
				}
			}
		}
	}
	else
	{
		if (transpose_B == false)
		{
			// Index of the first sub-matrix of A^T processed by the block
			int mat_A_t_begin = n*block_size*bid_x;
			
			// Step size used to iterate through the sub-matrices of A^T
			int mat_A_t_step  = block_size;
			
			// Index of the first sub-matrix of B processed by the block
			int mat_B_begin = n*block_size*bid_y;
			
			// Index of the last sub-matrix of B processed by the blcok
			int mat_B_end   = mat_B_begin + n - 1;
			
			// Step size used to iterate through the sub-matrices of B
			int mat_B_step  = block_size;
			
			// sum is used to store the element of the block sub-matrix
			// that is computed by the thread
			double sum = 0.0;
			
			// Counter to record the current positions of column position
			// of sub-matrix in matrix A^T and row position of sub-matrix
			// in matrix B
			int idx_A_t_col = 0;
			int idx_B_row = 0; 
			
			// Loop over all the sub-matrices of A^T and B
			// required to compute the block sub-matrix
			for (int idx_A_t = mat_A_t_begin, idx_B = mat_B_begin;
				 idx_B <= mat_B_end;
				 idx_A_t += mat_A_t_step, idx_B += mat_B_step)
			{
				// Load the matrices from device memory to shared memory;
				// Each thread loads one element of each sub-matrix
				
				if (g_tid_x < m && tid_y + idx_A_t_col < n)
					mat_A_shared[tid_x][tid_y] = d_mat_A[idx_A_t + n*tid_x + tid_y];
				if (tid_x + idx_B_row < n && g_tid_y < l)
					mat_B_shared[tid_x][tid_y] = d_mat_B[idx_B + n*tid_y + tid_x];
				
				// Synchronize to make sure the matrices are loaded
				__syncthreads();
				
				if (g_tid_x < m && g_tid_y < l)
				{
					int k_bound = min(block_size, n - idx_A_t_col);
					
					for (int k = 0; k < k_bound; k++)
					{
						sum += mat_A_shared[tid_x][k]*mat_B_shared[k][tid_y];
					}
				}
				
				idx_A_t_col += block_size;
				idx_B_row += block_size;
				
				// Synchronize to make sure that the preceding computation
				// is done before loading two new sub-matrices of A^T and B
				// in the next iteration
				__syncthreads();
			}
			
			// Write the block sub-matrix to device memory
			// each thread writes one element
			if (g_tid_x < m && g_tid_y < l)
			{
				int idx_D = m*block_size*bid_y + block_size*bid_x + tid_x + m*tid_y;
				
				if (beta == 0.0)
				{
					d_mat_D[idx_D] = alpha*sum;
				}
				else
				{
					d_mat_D[idx_D] = alpha*sum + beta*d_mat_C[idx_D];
				}
			}
		}
		else
		{
			// Index of the first sub-matrix of A^T processed by the block
			int mat_A_t_begin = n*block_size*bid_x;
			
			// Step size used to iterate through the sub-matrices of A^T
			int mat_A_t_step  = block_size;
			
			// Index of the first sub-matrix of B^T processed by the block
			int mat_B_t_begin = block_size*bid_y;
			
			// Index of the last sub-matrix of B^T processed by the blcok
			int mat_B_t_end   = mat_B_t_begin + (n - 1)*l;
			
			// Step size used to iterate through the sub-matrices of B^T
			int mat_B_t_step  = block_size*l;
			
			// sum is used to store the element of the block sub-matrix
			// that is computed by the thread
			double sum = 0.0;
			
			// Counter to record the current positions of column position
			// of sub-matrix in matrix A^T and row position of sub-matrix
			// in matrix B^T
			int idx_A_t_col = 0;
			int idx_B_t_row = 0; 
			
			// Loop over all the sub-matrices of A^T and B^T
			// required to compute the block sub-matrix
			for (int idx_A_t = mat_A_t_begin, idx_B_t = mat_B_t_begin;
				 idx_B_t <= mat_B_t_end;
				 idx_A_t += mat_A_t_step, idx_B_t += mat_B_t_step)
			{				
				// Load the matrices from device memory to shared memory;
				// Each thread loads one element of each sub-matrix
				
				if (g_tid_x < m && tid_y + idx_A_t_col < n)
					mat_A_shared[tid_x][tid_y] = d_mat_A[idx_A_t + n*tid_x + tid_y];
				if (tid_x + idx_B_t_row < n && g_tid_y < l)
					mat_B_shared[tid_x][tid_y] = d_mat_B[idx_B_t + l*tid_x + tid_y];
				
				// Synchronize to make sure the matrices are loaded
				__syncthreads();
				
				if (g_tid_x < m && g_tid_y < l)
				{
					int k_bound = min(block_size, n - idx_A_t_col);
					
					for (int k = 0; k < k_bound; k++)
					{
						sum += mat_A_shared[tid_x][k]*mat_B_shared[k][tid_y];
					}
				}
				
				idx_A_t_col += block_size;
				idx_B_t_row += block_size;
				
				// Synchronize to make sure that the preceding computation
				// is done before loading two new sub-matrices of A^T and B^T
				// in the next iteration
				__syncthreads();
			}
			
			// Write the block sub-matrix to device memory
			// each thread writes one element
			if (g_tid_x < m && g_tid_y < l)
			{
				int idx_D = m*block_size*bid_y + block_size*bid_x + tid_x + m*tid_y;
				
				if (beta == 0.0)
				{
					d_mat_D[idx_D] = alpha*sum;
				}
				else
				{
					d_mat_D[idx_D] = alpha*sum + beta*d_mat_C[idx_D];
				}
			}
		}
	}
}

/*
 * Algorithm 2 of general matrix-matrix multiplication (GEMM)
 * GEMM operation is expressed as D = alpha*op(A)*op(B) + beta*C
 * Blocking algorithm and shared memory is used in this algorithm
 * 
 * Parameters:
 *  m:              Number of rows of op(A) / number of rows of C/D
 *  n:              Number of columns of op(A) / number of rows of op(B)
 *  l:              Number of columns of op(B) / number of columns of C/D
 *  transpose_A:    Whether A should be transposed
 *                  If transpose_A is false, op(A) = A
 *                  Otherwise, op(A) = A^T
 *  transpose_B:    Whether B should be transposed
 *                  If transpose_B is false, op(B) = B
 *                  Otherwise, op(B) = B^T
 */
void gpu_GEMM_2 (const double alpha,
                 const double beta,
                 const double* const mat_A,
                 const double* const mat_B,
                 const double* const mat_C,
                 double* mat_D,
			     const int m,
			     const int n,
			     const int l,
				 const bool transpose_A,
				 const bool transpose_B)
{
	double *d_mat_A;
	double *d_mat_B;
	double *d_mat_C;
	double *d_mat_D;
	
	// Allocate the device memory
	checkCudaErrors(hipMalloc(&d_mat_A, m*n*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_B, n*l*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_C, m*l*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_D, m*l*sizeof(double)));
	
	// Copy data from the host memory to the device memory
	checkCudaErrors(hipMemcpy(d_mat_A, mat_A, m*n*sizeof(double), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_mat_B, mat_B, n*l*sizeof(double), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_mat_C, mat_C, m*l*sizeof(double), hipMemcpyHostToDevice));
	
	dim3 n_threads(0, 0);
	dim3 n_blocks(0, 0);
	
	// Set the size of the sub-block
	n_threads.x = BLOCK_SIZE_2;
	n_threads.y = BLOCK_SIZE_2;
	
	// Assume each dimension of the block is less than 65536
	// and compute the grid size
	n_blocks.x = (m + n_threads.x - 1)/n_threads.x;
	n_blocks.y = (l + n_threads.y - 1)/n_threads.y;
	
	// Launch the kernel
	device_GEMM_2 <BLOCK_SIZE_2> <<<n_blocks, n_threads>>> (alpha,
														    beta,
														    d_mat_A,
														    d_mat_B,
														    d_mat_C,
														    d_mat_D,
														    m,
														    n,
														    l,
														    transpose_A,
														    transpose_B);
	
	// Copy data from the device memory to the host memory
	checkCudaErrors(hipMemcpy(mat_D, d_mat_D, m*l*sizeof(double), hipMemcpyDeviceToHost));
	
	// Free the device memory
	hipFree(d_mat_A);
	hipFree(d_mat_B);
	hipFree(d_mat_C);
	hipFree(d_mat_D);
}

// Kernel to compute the third GEMM algorithm
template<int block_size_x, int block_size_y>
__global__
void device_GEMM_3 (const double alpha,
				    const double beta,
				    const double* const d_mat_A,
				    const double* const d_mat_B,
				    const double* const d_mat_C,
				    double* d_mat_D,
				    const int m,
				    const int n,
				    const int l,
					const bool transpose_A,
					const bool transpose_B)
{
	// Local thread index
	const int tid_x = threadIdx.x;
	const int tid_y = threadIdx.y;
	
	// Block index
	const int bid_x = blockIdx.x;
	const int bid_y = blockIdx.y;
	
	// Declare the shared memory to store sub-block of matrix B
	__shared__ double mat_B_shared[block_size_x][block_size_x+1];
	
	// Declare a local array in register to store the sums in the
	// elements the thread is responsible for
	double c[block_size_x];
	
#pragma unroll
	for (int i = 0; i< block_size_x; i++)
	{
		c[i] = 0.0;
	}
	
	if (transpose_A == false && transpose_B == false)
	{
		int idx_A = bid_x*block_size_x*block_size_y + tid_x + tid_y*block_size_x;
		int idx_B = tid_x + (bid_y*block_size_x + tid_y)*n;
		
		int idx_B_last = idx_B + n;
		int col_A = 0;
		
		do
		{
#pragma unroll
			for(int i = 0; i < block_size_x; i += block_size_y)
				mat_B_shared[tid_x][tid_y + i] = d_mat_B[idx_B + i*n];
			
			idx_B += block_size_x;
			
			__syncthreads();
			
			int i_bound = min(block_size_x, n - col_A);
			for (int i = 0; i < i_bound; i++, idx_A+=m)
			{
#pragma unroll
				for (int j = 0; j < block_size_x; j++)
				{
					c[j] += d_mat_A[idx_A]*mat_B_shared[i][j];
				}
			}
			
			col_A += block_size_x;
			
			__syncthreads();
		
		}while (idx_B < idx_B_last);
	}
	else if (transpose_A == false && transpose_B == true)
	{
		int idx_A = bid_x*block_size_x*block_size_y + tid_x + tid_y*block_size_x;
		int idx_B = tid_x*l + (bid_y*block_size_x + tid_y);
		
		int idx_B_last = idx_B + n*l;
		int col_A = 0;
		
		do
		{
#pragma unroll
			for(int i = 0; i < block_size_x; i += block_size_y)
				mat_B_shared[tid_x][tid_y + i] = d_mat_B[idx_B + i];
			
			idx_B += block_size_x*l;
			
			__syncthreads();
			
			int i_bound = min(block_size_x, n - col_A);
			for (int i = 0; i < i_bound; i++, idx_A+=m)
			{
#pragma unroll
				for (int j = 0; j < block_size_x; j++)
				{
					c[j] += d_mat_A[idx_A]*mat_B_shared[i][j];
				}
			}
			
			col_A += block_size_x;
			
			__syncthreads();
		
		}while (idx_B < idx_B_last);
	}
	else if (transpose_A == true && transpose_B == false)
	{
		int idx_A = (bid_x*block_size_x*block_size_y + tid_x + tid_y*block_size_x)*n;
		int idx_B = tid_x + (bid_y*block_size_x + tid_y)*n;

		int idx_B_last = idx_B + n;
		int col_A = 0;
		
		do
		{
#pragma unroll
			for(int i = 0; i < block_size_x; i += block_size_y)
				mat_B_shared[tid_x][tid_y + i] = d_mat_B[idx_B + i*n];
			
			idx_B += block_size_x;
			
			__syncthreads();
			
			int i_bound = min(block_size_x, n - col_A);
			for (int i = 0; i < i_bound; i++, idx_A++)
			{
#pragma unroll
				for (int j = 0; j < block_size_x; j++)
				{
					c[j] += d_mat_A[idx_A]*mat_B_shared[i][j];
				}
			}
			
			col_A += block_size_x;
			
			__syncthreads();
		
		}while (idx_B < idx_B_last);
	}
	else
	{
		int idx_A = (bid_x*block_size_x*block_size_y + tid_x + tid_y*block_size_x)*n;
		int idx_B = tid_x*l + (bid_y*block_size_x + tid_y);
		
		int idx_B_last = idx_B + n*l;
		int col_A = 0;
		do
		{
#pragma unroll
			for(int i = 0; i < block_size_x; i += block_size_y)
				mat_B_shared[tid_x][tid_y + i] = d_mat_B[idx_B + i];
			
			idx_B += block_size_x*l;
			
			__syncthreads();
			
			int i_bound = min(block_size_x, n - col_A);
			for (int i = 0; i < i_bound; i++, idx_A++)
			{
#pragma unroll
				for (int j = 0; j < block_size_x; j++)
				{
					c[j] += d_mat_A[idx_A]*mat_B_shared[i][j];
				}
			}
			
			col_A += block_size_x;
			
			__syncthreads();
		
		}while (idx_B < idx_B_last);
	}
	
	if (bid_x*block_size_x*block_size_y + tid_x + tid_y*block_size_x < m)
	{
		int idx_D = bid_x*block_size_x*block_size_y + (tid_x + tid_y*block_size_x) + bid_y*block_size_x*m;
		if (beta == 0.0)
		{
			int i_bound = min(block_size_x, l - bid_y*block_size_x);
			for (int i = 0; i < i_bound; i++, idx_D += m)
			{
				d_mat_D[idx_D] = alpha*c[i];
			}
		
		}
		else
		{
			int i_bound = min(block_size_x, l - bid_y*block_size_x);
			int idx_C = bid_x*block_size_x*block_size_y + (tid_x + tid_y*block_size_x) + bid_y*block_size_x*m;
			for (int i = 0; i < i_bound; i++, idx_C += m, idx_D += m)
			{
				d_mat_D[idx_D] = alpha*c[i] + beta*d_mat_C[idx_C];
			}
		}
	}
}

/*
 * Algorithm 3 of general matrix-matrix multiplication (GEMM)
 * GEMM operation is expressed as D = alpha*op(A)*op(B) + beta*C
 * A better blocking algorithm and shared memory is used in this algorithm
 * 
 * Parameters:
 *  m:              Number of rows of op(A) / number of rows of C/D
 *  n:              Number of columns of op(A) / number of rows of op(B)
 *  l:              Number of columns of op(B) / number of columns of C/D
 *  transpose_A:    Whether A should be transposed
 *                  If transpose_A is false, op(A) = A
 *                  Otherwise, op(A) = A^T
 *  transpose_B:    Whether B should be transposed
 *                  If transpose_B is false, op(B) = B
 *                  Otherwise, op(B) = B^T
 */
void gpu_GEMM_3 (const double alpha,
                 const double beta,
                 const double* const mat_A,
                 const double* const mat_B,
                 const double* const mat_C,
                 double* mat_D,
			     const int m,
			     const int n,
			     const int l,
				 const bool transpose_A,
				 const bool transpose_B)
{
	double *d_mat_A;
	double *d_mat_B;
	double *d_mat_C;
	double *d_mat_D;
	
	// Allocate the device memory
	checkCudaErrors(hipMalloc(&d_mat_A, m*n*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_B, n*l*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_C, m*l*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_D, m*l*sizeof(double)));
	
	// Copy data from the host memory to the device memory
	checkCudaErrors(hipMemcpy(d_mat_A, mat_A, m*n*sizeof(double), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_mat_B, mat_B, n*l*sizeof(double), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_mat_C, mat_C, m*l*sizeof(double), hipMemcpyHostToDevice));
	
	dim3 n_threads(0, 0);
	dim3 n_blocks(0, 0);
	
	// Set the size of each block
	n_threads.x = BLOCK_SIZE_x_3;
	n_threads.y = BLOCK_SIZE_y_3;
	
	// Assume each dimension of the block is less than 65536
	// and compute the grid size
	n_blocks.x = (m + n_threads.x*n_threads.y - 1)/(n_threads.x*n_threads.y);
	n_blocks.y = (l + n_threads.x - 1)/n_threads.x;
		
	// Launch the kernel
	device_GEMM_3 <BLOCK_SIZE_x_3, BLOCK_SIZE_y_3> <<<n_blocks, n_threads>>> (alpha,
																		      beta,
																		      d_mat_A,
																		      d_mat_B,
																		      d_mat_C,
																		      d_mat_D,
																		      m,
																		      n,
																		      l,
																		      transpose_A,
																		      transpose_B);
	
	// Copy data from the device memory to the host memory
	checkCudaErrors(hipMemcpy(mat_D, d_mat_D, m*l*sizeof(double), hipMemcpyDeviceToHost));
	
	// Free the device memory
	hipFree(d_mat_A);
	hipFree(d_mat_B);
	hipFree(d_mat_C);
	hipFree(d_mat_D);
}

// Kernel to compute the sigmoid function
__global__
void device_sigmoid (const double* const d_mat_1,
				     double* d_mat_2,
				     const int m,
				     const int n)
{
	// Global thread index in the grid
	const int tid_x = threadIdx.x + blockDim.x*blockIdx.x;
	const int tid_y = threadIdx.y + blockDim.y*blockIdx.y;
	
	// If the thread is not inside the matrix D, return
	if (tid_x >= m || tid_y >= n)
	{
		return;
	}
	
	// Compute the index in the matrix
	int idx = tid_y*m + tid_x;
	
	// Apply the sigmoid function
	d_mat_2[idx] = 1.0 / (1.0 + exp(-d_mat_1[idx]));
}

/*
 * Applies the sigmoid function to each element of the matrix
 * and returns a new matrix by GPU
 *  m: number of rows of the matrix
 *  n: number of columns of the matrix
 */
void gpu_sigmoid (const double* const mat_1,
                  double* mat_2,
                  const int m,
                  const int n)
{
	double *d_mat_1;
	double *d_mat_2;
	
	// Allocate the device memory
	checkCudaErrors(hipMalloc(&d_mat_1, m*n*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_2, m*n*sizeof(double)));
	
	// Copy data from the host memory to the device memory
	checkCudaErrors(hipMemcpy(d_mat_1, mat_1, m*n*sizeof(double), hipMemcpyHostToDevice));
	
	dim3 n_threads(0, 0);
	dim3 n_blocks(0, 0);
	
	// Compute the block dimension
	n_threads.x = BLOCK_SIZE_x_SIGMOID;
	n_threads.y = BLOCK_SIZE_y_SIGMOID;
	
	// Compute the grid size
	n_blocks.x = (m + n_threads.x - 1)/n_threads.x;
	n_blocks.y = (n + n_threads.y - 1)/n_threads.y;
	
	// Launch the kernel to apply the sigmoid function
	device_sigmoid <<<n_blocks, n_threads>>> (d_mat_1, d_mat_2, m, n);
	
	// Copy data from the device memory to the host memory
	checkCudaErrors(hipMemcpy(mat_2, d_mat_2, m*n*sizeof(double), hipMemcpyDeviceToHost));
	
	// Free the device memory
	hipFree(d_mat_1);
	hipFree(d_mat_2);
}

// Kernel to compute the exponential function
__global__
void device_exponent (const double* const d_mat_1,
				      double* d_mat_2,
				      const int m,
				      const int n)
{
	// Global thread index in the grid
	const int tid_x = threadIdx.x + blockDim.x*blockIdx.x;
	const int tid_y = threadIdx.y + blockDim.y*blockIdx.y;
	
	// If the thread is not inside the matrix D, return
	if (tid_x >= m || tid_y >= n)
	{
		return;
	}
	
	// Compute the index in the matrix
	int idx = tid_y*m + tid_x;
	
	// Apply the exponential function
	d_mat_2[idx] = exp(d_mat_1[idx]);
}

__device__
int nextPowerOf2(const int x)
{
	return (1 << (32 - __clz(x - 1)));
}

// Kernel to do reduction along rows
template<int block_size>
__global__
void device_sum_row_block(const double* const d_mat,
						  double* d_col_vec,
						  const int m,
						  const int n)
{	
	// Local thead index and block index
	const int tid_y = threadIdx.y;
	const int bid_x = blockIdx.x;
	
	double my_sum = 0.0;
	for (int i = tid_y; i < n; i += block_size)
	{
		// Compute the index in the matrix
		int idx = i*m + (threadIdx.x + blockDim.x*blockIdx.x);
		my_sum += d_mat[idx];
	}
	
	__shared__ double smem[block_size];
	
	smem[tid_y] = my_sum;
	
	__syncthreads();
	
	//use this for non-power of 2 block_sizes
	for (int shift = nextPowerOf2(block_size) / 2;
		 shift > 0;
		 shift >>= 1)
	{
		if (tid_y < shift && tid_y + shift < block_size)
		{
			smem[tid_y] += smem[tid_y + shift];
		}
		__syncthreads();
	}
	
	if (tid_y == 0)
		d_col_vec[bid_x] = smem[tid_y];
}

// Kernel to normalize the outpus after applying exponential function
__global__
void device_normalize (double* d_mat,
					   const double* d_col_vec,
					   const int m,
					   const int n)
{
	// Global thread index in the grid
	const int tid_x = threadIdx.x + blockDim.x*blockIdx.x;
	const int tid_y = threadIdx.y + blockDim.y*blockIdx.y;
	
	// If the thread is not inside the matrix D, return
	if (tid_x >= m || tid_y >= n)
	{
		return;
	}
	
	// Compute the index in the matrix
	int idx = tid_y*m + tid_x;
	
	// Apply the exponential function
	d_mat[idx] = d_mat[idx]/d_col_vec[tid_x];
}

/*
 * Applies the softmax to each rowvec of the matrix by GPU
 *  m: number of rows of the matrix
 *  n: number of columns of the matrix
 */
void gpu_softmax (const double* const mat_1,
                  double* mat_2,
                  const int m,
                  const int n)
{
	double *d_mat_1;
	double *d_mat_2;
	double *d_col_vec;

	// Allocate the device memory
	checkCudaErrors(hipMalloc(&d_mat_1, m*n*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_2, m*n*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_col_vec, m*sizeof(double)));
	
	// Copy data from the host memory to the device memory
	checkCudaErrors(hipMemcpy(d_mat_1, mat_1, m*n*sizeof(double), hipMemcpyHostToDevice));
	
	dim3 n_threads(0, 0);
	dim3 n_blocks(0, 0);
	
	// Compute the block dimension
	n_threads.x = BLOCK_SIZE_x_SOFTMAX;
	n_threads.y = BLOCK_SIZE_y_SOFTMAX;
	
	// Compute the grid size
	n_blocks.x = (m + n_threads.x - 1)/n_threads.x;
	n_blocks.y = (n + n_threads.y - 1)/n_threads.y;
	
	// Launch the kernel to compute the elementwise exponentinal function
	device_exponent <<<n_blocks, n_threads>>> (d_mat_1, d_mat_2, m, n);
	
	// Compute the block dimension
	n_threads.x = 1;
	n_threads.y = BLOCK_SIZE_REDUCTION;
	
	// Compute the grid size
	n_blocks.x = m;
	n_blocks.y = 1;
	
	device_sum_row_block <BLOCK_SIZE_REDUCTION> <<<n_blocks, n_threads>>> (d_mat_2, d_col_vec, m, n);
	
	// Compute the block dimension
	n_threads.x = BLOCK_SIZE_x_SOFTMAX;
	n_threads.y = BLOCK_SIZE_y_SOFTMAX;
	
	// Compute the grid size
	n_blocks.x = (m + n_threads.x - 1)/n_threads.x;
	n_blocks.y = (n + n_threads.y - 1)/n_threads.y;
	
	device_normalize <<<n_blocks, n_threads>>> (d_mat_2, d_col_vec, m, n);
	
	// Copy data from the device memory to the host memory
	checkCudaErrors(hipMemcpy(mat_2, d_mat_2, m*n*sizeof(double), hipMemcpyDeviceToHost));

	// Free the device memory
	hipFree(d_mat_1);
	hipFree(d_mat_2);
	hipFree(d_col_vec);
}

// Kernel to do reduction along columns
template<int block_size>
__global__
void device_sum_col_block(const double* const d_mat,
						  double* d_row_vec,
						  const int m,
						  const int n)
{
	// Local thead index and block index
	const int tid_x = threadIdx.x;
	const int bid_y = blockIdx.y;
	
	double my_sum = 0.0;
	for (int i = tid_x; i < m; i += block_size)
	{
		// Compute the index in the matrix
		int idx = (threadIdx.y + blockDim.y*blockIdx.y)*m + i;
		my_sum += d_mat[idx];
	}
	
	__shared__ double smem[block_size];
	
	smem[tid_x] = my_sum;

	__syncthreads();
	
	//use this for non-power of 2 block_sizes
	for (int shift = nextPowerOf2(block_size) / 2;
		 shift > 0;
		 shift >>= 1)
	{
		if (tid_x < shift && tid_x + shift < block_size)
		{
			smem[tid_x] += smem[tid_x + shift];
		}
		__syncthreads();
	}

	if (tid_x == 0)
		d_row_vec[bid_y] = smem[tid_x];
}

/*
 * Sum elements of matrix in each column
 *  m: number of rows of the matrix
 *  n: number of columns of the matrix
 */
void gpu_sum_col (const double* const mat,
                  double* row_vec,
                  const int m,
                  const int n)
{
	double *d_mat;
	double *d_row_vec;

	// Allocate the device memory
	checkCudaErrors(hipMalloc(&d_mat, m*n*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_row_vec, n*sizeof(double)));
	
	// Copy data from the host memory to the device memory
	checkCudaErrors(hipMemcpy(d_mat, mat, m*n*sizeof(double), hipMemcpyHostToDevice));
	
	dim3 n_threads(0, 0);
	dim3 n_blocks(0, 0);
	
	// Compute the block dimension
	n_threads.x = BLOCK_SIZE_REDUCTION;
	n_threads.y = 1;
	
	// Compute the grid size
	n_blocks.x = 1;
	n_blocks.y = n;
	
	device_sum_col_block <BLOCK_SIZE_REDUCTION> <<<n_blocks, n_threads>>> (d_mat, d_row_vec, m, n);
	
	// Copy data from the device memory to the host memory
	checkCudaErrors(hipMemcpy(row_vec, d_row_vec, n*sizeof(double), hipMemcpyDeviceToHost));

	// Free the device memory
	hipFree(d_mat);
	hipFree(d_row_vec);
}

// Kernel to compute the elementwise multiplication in computing dW1
__global__
void device_elementwise_mult (const double* const d_mat_da1,
							  const double* const d_mat_a1,
							  double* d_mat_dz1,
							  const int m,
							  const int n)
{
	// Global thread index in the grid
	const int tid_x = threadIdx.x + blockDim.x*blockIdx.x;
	const int tid_y = threadIdx.y + blockDim.y*blockIdx.y;
	
	// If the thread is not inside the matrix D, return
	if (tid_x >= m || tid_y >= n)
	{
		return;
	}
	
	// Compute the index in the matrix
	int idx = tid_y*m + tid_x;
	
	// Apply the sigmoid function
	d_mat_dz1[idx] = d_mat_da1[idx]*d_mat_a1[idx]*(1.0 - d_mat_a1[idx]);
}

/*
 * Elementwise multiplication used to compute dW1
 * and return a new matrix by GPU
 *  mat_da1: input matrix da1
 *  mat_a1:  input matrix a1
 *  mat_dz1: output matrix dz1
 *  m:     number of rows of the matrices
 *  n:     number of columns of the matrices
 */
void gpu_elementwise_mult (const double* const mat_da1,
						   const double* const mat_a1,
						   double* mat_dz1,
						   const int m,
						   const int n)
{
	double *d_mat_da1;
	double *d_mat_a1;
	double *d_mat_dz1;
	
	// Allocate the device memory
	checkCudaErrors(hipMalloc(&d_mat_da1, m*n*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_a1, m*n*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_dz1, m*n*sizeof(double)));
	
	// Copy data from the host memory to the device memory
	checkCudaErrors(hipMemcpy(d_mat_da1, mat_da1, m*n*sizeof(double), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_mat_a1, mat_a1, m*n*sizeof(double), hipMemcpyHostToDevice));
	
	dim3 n_threads(0, 0);
	dim3 n_blocks(0, 0);
	
	// Compute the block dimension
	n_threads.x = BLOCK_SIZE_x_elementwise_mult;
	n_threads.y = BLOCK_SIZE_y_elementwise_mult;
	
	// Compute the grid size
	n_blocks.x = (m + n_threads.x - 1)/n_threads.x;
	n_blocks.y = (n + n_threads.y - 1)/n_threads.y;
	
	// Launch the kernel to do the elementwise multiplication
	device_elementwise_mult <<<n_blocks, n_threads>>> (d_mat_da1, d_mat_a1, d_mat_dz1, m, n);
	
	// Copy data from the device memory to the host memory
	checkCudaErrors(hipMemcpy(mat_dz1, d_mat_dz1, m*n*sizeof(double), hipMemcpyDeviceToHost));
	
	// Free the device memory
	hipFree(d_mat_da1);
	hipFree(d_mat_a1);
	hipFree(d_mat_dz1);
}

// Kernel to compute the elementwise subtraction
// C = A - alpha * B
__global__
void device_elementwise_subtract (const double alpha,
								  const double* const d_mat_A,
								  const double* const d_mat_B,
								  double* d_mat_C,
								  const int m,
								  const int n)
{
	// Global thread index in the grid
	const int tid_x = threadIdx.x + blockDim.x*blockIdx.x;
	const int tid_y = threadIdx.y + blockDim.y*blockIdx.y;
	
	// If the thread is not inside the matrix D, return
	if (tid_x >= m || tid_y >= n)
	{
		return;
	}
	
	// Compute the index in the matrix
	int idx = tid_y*m + tid_x;
	
	// Apply the sigmoid function
	d_mat_C[idx] = d_mat_A[idx] - alpha*d_mat_B[idx];
}

/*
 * Elementwise substraction between two matrices by GPU
 * C = A - alpha * B
 * A new matrix is returned
 *  mat_A: input matrix A
 *  mat_B: input matrix B
 *  mat_C: output matrix C
 *  m:     number of rows of the matrices
 *  n:     number of columns of the matrices
 */
void gpu_elementwise_subtract (const double alpha,
							   const double* const mat_A,
							   const double* const mat_B,
							   double* mat_C,
							   const int m,
							   const int n)
{
	double *d_mat_A;
	double *d_mat_B;
	double *d_mat_C;
	
	// Allocate the device memory
	checkCudaErrors(hipMalloc(&d_mat_A, m*n*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_B, m*n*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_C, m*n*sizeof(double)));
	
	// Copy data from the host memory to the device memory
	checkCudaErrors(hipMemcpy(d_mat_A, mat_A, m*n*sizeof(double), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_mat_B, mat_B, m*n*sizeof(double), hipMemcpyHostToDevice));
	
	dim3 n_threads(0, 0);
	dim3 n_blocks(0, 0);
	
	// Compute the block dimension
	n_threads.x = BLOCK_SIZE_x_elementwise_subtract;
	n_threads.y = BLOCK_SIZE_y_elementwise_subtract;
	
	// Compute the grid size
	n_blocks.x = (m + n_threads.x - 1)/n_threads.x;
	n_blocks.y = (n + n_threads.y - 1)/n_threads.y;
	
	// Launch the kernel to do the elementwise substraction
	device_elementwise_subtract <<<n_blocks, n_threads>>> (alpha, d_mat_A, d_mat_B, d_mat_C, m, n);
	
	// Copy data from the device memory to the host memory
	checkCudaErrors(hipMemcpy(mat_C, d_mat_C, m*n*sizeof(double), hipMemcpyDeviceToHost));
	
	// Free the device memory
	hipFree(d_mat_A);
	hipFree(d_mat_B);
	hipFree(d_mat_C);
}

// Kernel to compute the diff matrix used in gpu_accel_feedforward_backprop()
__global__
void device_compute_diff (const double* const d_mat_yc,
						  const double* const d_mat_y,
						  double* d_mat_diff,
						  const int m,
						  const int n)
{
	// Global thread index in the grid
	const int tid_x = threadIdx.x + blockDim.x*blockIdx.x;
	const int tid_y = threadIdx.y + blockDim.y*blockIdx.y;
	
	// If the thread is not inside the matrix D, return
	if (tid_x >= m || tid_y >= n)
	{
		return;
	}
	
	// Compute the index in the matrix
	int idx = tid_y*m + tid_x;
	
	// Apply the difference function
	d_mat_diff[idx] = 1.0/((double) m) * (d_mat_yc[idx] - d_mat_y[idx]);
}

/*
 * Compute the diff matrix
 * and returns a new matrix by GPU
 *  mat_yc:   input matrix yc
 *  mat_y:    input matrix y
 *  mat_diff: output matrix diff
 *  m:     number of rows of the matrices
 *  n:     number of columns of the matrices
 */
void gpu_compute_diff (const double* const mat_yc,
					   const double* const mat_y,
					   double* mat_diff,
					   const int m,
					   const int n)
{
	double *d_mat_yc;
	double *d_mat_y;
	double *d_mat_diff;
	
	// Allocate the device memory
	checkCudaErrors(hipMalloc(&d_mat_yc, m*n*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_y, m*n*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_diff, m*n*sizeof(double)));
	
	// Copy data from the host memory to the device memory
	checkCudaErrors(hipMemcpy(d_mat_yc, mat_yc, m*n*sizeof(double), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_mat_y, mat_y, m*n*sizeof(double), hipMemcpyHostToDevice));
	
	dim3 n_threads(0, 0);
	dim3 n_blocks(0, 0);
	
	// Compute the block dimension
	n_threads.x = BLOCK_SIZE_x_DIFF;
	n_threads.y = BLOCK_SIZE_y_DIFF;
	
	// Compute the grid size
	n_blocks.x = (m + n_threads.x - 1)/n_threads.x;
	n_blocks.y = (n + n_threads.y - 1)/n_threads.y;
	
	// Launch the kernel to apply the sigmoid function
	device_compute_diff <<<n_blocks, n_threads>>> (d_mat_yc, d_mat_y, d_mat_diff, m, n);
	
	// Copy data from the device memory to the host memory
	checkCudaErrors(hipMemcpy(mat_diff, d_mat_diff, m*n*sizeof(double), hipMemcpyDeviceToHost));
	
	// Free the device memory
	hipFree(d_mat_yc);
	hipFree(d_mat_y);
	hipFree(d_mat_diff);
}

// Kernel to transpose a matrix
__global__
void device_transpose (double* d_mat_in, double* d_mat_out, const int m, const int n)
{
    const int row = threadIdx.x + blockDim.x * blockIdx.x;
    const int col = threadIdx.y + blockDim.y * blockIdx.y;
	
	if (row >= m || col >= n)
	{
		return;
	}
	
	const int idx_in = col*m + row;
	const int idx_out = row*n + col;
	
    d_mat_out[idx_out] = d_mat_in[idx_in];
}

/*
 * Transpose a matrix and return a new matrix by GPU
 *  mat_1: input matrix 
 *  mat_2: output matrix
 *  m:     number of rows of the input matrix
 *  n:     number of columns of the input matrix
 */
void gpu_transpose (const double* const mat_1,
					double* mat_2,
					const int m,
					const int n)
{
	double *d_mat_1;
	double *d_mat_2;
	
	// Allocate the device memory
	checkCudaErrors(hipMalloc(&d_mat_1, m*n*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_2, m*n*sizeof(double)));
	
	// Copy data from the host memory to the device memory
	checkCudaErrors(hipMemcpy(d_mat_1, mat_1, m*n*sizeof(double), hipMemcpyHostToDevice));
	
	dim3 n_threads(0, 0);
	dim3 n_blocks(0, 0);
	
	// Compute the block dimension
	n_threads.x = BLOCK_SIZE_x_transpose;
	n_threads.y = BLOCK_SIZE_y_transpose;
	
	// Compute the grid size
	n_blocks.x = (m + n_threads.x - 1)/n_threads.x;
	n_blocks.y = (n + n_threads.y - 1)/n_threads.y;
	
	// Launch the kernel to do the transpose
	device_transpose <<<n_blocks, n_threads>>> (d_mat_1, d_mat_2, m, n);
	
	// Copy data from the device memory to the host memory
	checkCudaErrors(hipMemcpy(mat_2, d_mat_2, m*n*sizeof(double), hipMemcpyDeviceToHost));
	
	// Free the device memory
	hipFree(d_mat_1);
	hipFree(d_mat_2);
}

/*
 * Do the feedforward in GPU entirely
 */
void gpu_accel_feedforward (const double* const mat_X, int X_n_rows, int X_n_cols,
                            const double* const mat_W1, int W1_n_rows, int W1_n_cols,
                            const double* const mat_b1, int b1_n_rows, int b1_n_cols,
                            double* mat_z1, int z1_n_rows, int z1_n_cols,
                            double* mat_a1, int a1_n_rows, int a1_n_cols,
                            const double* const mat_W2, int W2_n_rows, int W2_n_cols,
                            const double* const mat_b2, int b2_n_rows, int b2_n_cols,
                            double* mat_z2, int z2_n_rows, int z2_n_cols,
                            double* mat_a2, int a2_n_rows, int a2_n_cols)
{
	double* d_mat_X;
	double* d_mat_W1;
	double* d_mat_b1;
	double* d_mat_z1;
	double* d_mat_a1;
	double* d_mat_W2;
	double* d_mat_b2;
	double* d_mat_z2;
	double* d_mat_a2;
	
	dim3 n_threads(0, 0);
	dim3 n_blocks(0, 0);
	
	// Allocate the device memory
	checkCudaErrors(hipMalloc(&d_mat_X, X_n_rows*X_n_cols*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_W1, W1_n_rows*W1_n_cols*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_b1, b1_n_rows*b1_n_cols*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_z1, z1_n_rows*z1_n_cols*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_a1, a1_n_rows*a1_n_cols*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_W2, W2_n_rows*W2_n_cols*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_b2, b2_n_rows*b2_n_cols*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_z2, z2_n_rows*z2_n_cols*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_a2, a2_n_rows*a2_n_cols*sizeof(double)));
	
	// Copy data from the host memory to the device memory
	checkCudaErrors(hipMemcpy(d_mat_X, mat_X, X_n_rows*X_n_cols*sizeof(double), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_mat_W1, mat_W1, W1_n_rows*W1_n_cols*sizeof(double), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_mat_b1, mat_b1, b1_n_rows*b1_n_cols*sizeof(double), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_mat_W2, mat_W2, W2_n_rows*W2_n_cols*sizeof(double), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_mat_b2, mat_b2, b2_n_rows*b2_n_cols*sizeof(double), hipMemcpyHostToDevice));
	
	/*
	 * Do the feedforward for z1
	 */
	
	// Set the size of the sub-block
	n_threads.x = BLOCK_SIZE_2;
	n_threads.y = BLOCK_SIZE_2;
	
	// Assume each dimension of the block is less than 65536
	// and compute the grid size
	n_blocks.x = (z1_n_rows + n_threads.x - 1)/n_threads.x;
	n_blocks.y = (z1_n_cols + n_threads.y - 1)/n_threads.y;
	
	// Launch the kernel to do GEMM
	device_GEMM_2 <BLOCK_SIZE_2> <<<n_blocks, n_threads>>> (1.0,
														    1.0,
														    d_mat_X,
														    d_mat_W1,
														    d_mat_b1,
														    d_mat_z1,
														    X_n_rows,
														    X_n_cols,
														    W1_n_rows,
														    false,
														    true);
		
	/*
	 * Compute the sigmoid of z1
	 */
	
	// Compute the block dimension
	n_threads.x = BLOCK_SIZE_x_SIGMOID;
	n_threads.y = BLOCK_SIZE_y_SIGMOID;
	
	// Compute the grid size
	n_blocks.x = (z1_n_rows + n_threads.x - 1)/n_threads.x;
	n_blocks.y = (z1_n_cols + n_threads.y - 1)/n_threads.y;
	
	// Launch the kernel to apply the sigmoid function
	device_sigmoid <<<n_blocks, n_threads>>> (d_mat_z1, d_mat_a1, z1_n_rows, z1_n_cols);
	
	/*
	 * Do the feedforward for z2
	 */
	
	// Set the size of the sub-block
	n_threads.x = BLOCK_SIZE_2;
	n_threads.y = BLOCK_SIZE_2;
	
	// Assume each dimension of the block is less than 65536
	// and compute the grid size
	n_blocks.x = (z2_n_rows + n_threads.x - 1)/n_threads.x;
	n_blocks.y = (z2_n_cols + n_threads.y - 1)/n_threads.y;
	
	// Launch the kernel to do GEMM
	device_GEMM_2 <BLOCK_SIZE_2> <<<n_blocks, n_threads>>> (1.0,
														    1.0,
														    d_mat_a1,
														    d_mat_W2,
														    d_mat_b2,
														    d_mat_z2,
														    a1_n_rows,
														    a1_n_cols,
														    W2_n_rows,
														    false,
														    true);
	
	/*
	 * Compute the softmax of z2
	 */
	
	double* d_col_vec;
	checkCudaErrors(hipMalloc(&d_col_vec, z2_n_rows*sizeof(double)));
	
	// Compute the block dimension
	n_threads.x = BLOCK_SIZE_x_SOFTMAX;
	n_threads.y = BLOCK_SIZE_y_SOFTMAX;
	
	// Compute the grid size
	n_blocks.x = (z2_n_rows + n_threads.x - 1)/n_threads.x;
	n_blocks.y = (z2_n_cols + n_threads.y - 1)/n_threads.y;
	
	// Launch the kernel to compute the elementwise exponentinal function
	device_exponent <<<n_blocks, n_threads>>> (d_mat_z2, d_mat_a2, z2_n_rows, z2_n_cols);
	
	// Compute the block dimension
	n_threads.x = 1;
	n_threads.y = BLOCK_SIZE_REDUCTION;
	
	// Compute the grid size
	n_blocks.x = a2_n_rows;
	n_blocks.y = 1;
	
	device_sum_row_block <BLOCK_SIZE_REDUCTION> <<<n_blocks, n_threads>>> (d_mat_a2, d_col_vec, a2_n_rows, a2_n_cols);
	
	// Compute the block dimension
	n_threads.x = BLOCK_SIZE_x_SOFTMAX;
	n_threads.y = BLOCK_SIZE_y_SOFTMAX;
	
	// Compute the grid size
	n_blocks.x = (a2_n_rows + n_threads.x - 1)/n_threads.x;
	n_blocks.y = (a2_n_cols + n_threads.y - 1)/n_threads.y;
	
	device_normalize <<<n_blocks, n_threads>>> (d_mat_a2, d_col_vec, a2_n_rows, a2_n_cols);
	
	hipFree(d_col_vec);
	
	// Copy data from the device memory to the host memory
	checkCudaErrors(hipMemcpy(mat_z1, d_mat_z1, z1_n_rows*z1_n_cols*sizeof(double), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(mat_a1, d_mat_a1, a1_n_rows*a1_n_cols*sizeof(double), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(mat_z2, d_mat_z2, z2_n_rows*z2_n_cols*sizeof(double), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(mat_a2, d_mat_a2, a2_n_rows*a2_n_cols*sizeof(double), hipMemcpyDeviceToHost));
		
	// Free the device memory
	hipFree(d_mat_X);
	hipFree(d_mat_W1);
	hipFree(d_mat_b1);
	hipFree(d_mat_z1);
	hipFree(d_mat_a1);
	hipFree(d_mat_W2);
	hipFree(d_mat_b2);
	hipFree(d_mat_z2);
	hipFree(d_mat_a2);
}

/*
 * Do the backpropagation in GPU entirely
 */
void gpu_accel_backprop (const double reg,
						 const double* const mat_diff, const int diff_n_rows, const int diff_n_cols,
                         const double* const mat_X, const int X_n_rows, const int X_n_cols,
                         const double* const mat_a1, const int a1_n_rows, const int a1_n_cols,
                         const double* const mat_W1, const int W1_n_rows, const int W1_n_cols,
                         const double* const mat_W2, const int W2_n_rows, const int W2_n_cols,
                         double* mat_dW1, const int dW1_n_rows, const int dW1_n_cols,
                         double* mat_dW2, const int dW2_n_rows, const int dW2_n_cols,
                         double* mat_db1, const int db1_n_cols,
                         double* mat_db2, const int db2_n_cols)
{
	double* d_mat_diff;
	double* d_mat_X;
	double* d_mat_a1;
	double* d_mat_W1;
	double* d_mat_W2;
	double* d_mat_dz1;
	double* d_mat_da1;
	double* d_mat_dW1;
	double* d_mat_dW2;
	double* d_mat_db1;
	double* d_mat_db2;
	
	dim3 n_threads(0, 0);
	dim3 n_blocks(0, 0);
	
	// Allocate the device memory
	checkCudaErrors(hipMalloc(&d_mat_diff, diff_n_rows*diff_n_cols*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_X, X_n_rows*X_n_cols*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_a1, a1_n_rows*a1_n_cols*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_W1, W1_n_rows*W1_n_cols*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_W2, W2_n_rows*W2_n_cols*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_dz1, diff_n_rows*W2_n_cols*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_da1, diff_n_rows*W2_n_cols*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_dW1, dW1_n_rows*dW1_n_cols*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_dW2, dW2_n_rows*dW2_n_cols*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_db1, db1_n_cols*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_db2, db2_n_cols*sizeof(double)));
	
	// Copy data from the host memory to the device memory
	checkCudaErrors(hipMemcpy(d_mat_diff, mat_diff, diff_n_rows*diff_n_cols*sizeof(double), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_mat_X, mat_X, X_n_rows*X_n_cols*sizeof(double), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_mat_a1, mat_a1, a1_n_rows*a1_n_cols*sizeof(double), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_mat_W1, mat_W1, W1_n_rows*W1_n_cols*sizeof(double), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_mat_W2, mat_W2, W2_n_rows*W2_n_cols*sizeof(double), hipMemcpyHostToDevice));
	
	/*
	 * Compute dW2
	 */
	
	// Set the size of the sub-block
	n_threads.x = BLOCK_SIZE_2;
	n_threads.y = BLOCK_SIZE_2;
	
	// Assume each dimension of the block is less than 65536
	// and compute the grid size
	n_blocks.x = (dW2_n_rows + n_threads.x - 1)/n_threads.x;
	n_blocks.y = (dW2_n_cols + n_threads.y - 1)/n_threads.y;
	
	// Launch the kernel to do GEMM
	device_GEMM_2 <BLOCK_SIZE_2> <<<n_blocks, n_threads>>> (1.0,
														    reg,
														    d_mat_diff,
														    d_mat_a1,
														    d_mat_W2,
														    d_mat_dW2,
														    diff_n_cols,
														    diff_n_rows,
														    a1_n_cols,
														    true,
														    false);
	
	/*
	 * Compute dW1
	 */
	
	n_threads.x = BLOCK_SIZE_2;
	n_threads.y = BLOCK_SIZE_2;
	
	// Assume each dimension of the block is less than 65536
	// and compute the grid size
	n_blocks.x = (diff_n_rows + n_threads.x - 1)/n_threads.x;
	n_blocks.y = (W2_n_cols + n_threads.y - 1)/n_threads.y;
	
	// Launch the kernel to do GEMM
	device_GEMM_2 <BLOCK_SIZE_2> <<<n_blocks, n_threads>>> (1.0,
														    0.0,
														    d_mat_diff,
														    d_mat_W2,
														    d_mat_da1,
														    d_mat_da1,
														    diff_n_rows,
														    diff_n_cols,
														    W2_n_cols,
														    false,
														    false);

	// Compute the block dimension
	n_threads.x = BLOCK_SIZE_x_elementwise_mult;
	n_threads.y = BLOCK_SIZE_y_elementwise_mult;
	
	// Compute the grid size
	n_blocks.x = (diff_n_rows + n_threads.x - 1)/n_threads.x;
	n_blocks.y = (W2_n_cols + n_threads.y - 1)/n_threads.y;
	
	// Launch the kernel to do the elementwise multiplication
	device_elementwise_mult <<<n_blocks, n_threads>>> (d_mat_da1, d_mat_a1, d_mat_dz1, diff_n_rows, W2_n_cols);
	
	n_threads.x = BLOCK_SIZE_2;
	n_threads.y = BLOCK_SIZE_2;
	
	// Assume each dimension of the block is less than 65536
	// and compute the grid size
	n_blocks.x = (W2_n_cols + n_threads.x - 1)/n_threads.x;
	n_blocks.y = (X_n_cols + n_threads.y - 1)/n_threads.y;
	
	// Launch the kernel to do GEMM
	device_GEMM_2 <BLOCK_SIZE_2> <<<n_blocks, n_threads>>> (1.0,
														    reg,
														    d_mat_dz1,
														    d_mat_X,
														    d_mat_W1,
														    d_mat_dW1,
														    W2_n_cols,
														    diff_n_rows,
														    X_n_cols,
														    true,
														    false);
	
	/*
	 * Compute db1
	 */
	
	// Compute the block dimension
	n_threads.x = BLOCK_SIZE_REDUCTION;
	n_threads.y = 1;
	
	// Compute the grid size
	n_blocks.x = 1;
	n_blocks.y = diff_n_rows;
	
	device_sum_col_block <BLOCK_SIZE_REDUCTION> <<<n_blocks, n_threads>>> (d_mat_diff, d_mat_db2, diff_n_cols, diff_n_rows);
	
	/*
	 * Compute db2
	 */
	
	// Compute the block dimension
	n_threads.x = BLOCK_SIZE_REDUCTION;
	n_threads.y = 1;
	
	// Compute the grid size
	n_blocks.x = 1;
	n_blocks.y = W2_n_cols;
	
	device_sum_col_block <BLOCK_SIZE_REDUCTION> <<<n_blocks, n_threads>>> (d_mat_dz1, d_mat_db1, diff_n_rows, W2_n_cols);
	
	// Copy data from the device memory to the host memory
	checkCudaErrors(hipMemcpy(mat_dW1, d_mat_dW1, dW1_n_rows*dW1_n_cols*sizeof(double), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(mat_dW2, d_mat_dW2, dW2_n_rows*dW2_n_cols*sizeof(double), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(mat_db1, d_mat_db1, db1_n_cols*sizeof(double), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(mat_db2, d_mat_db2, db2_n_cols*sizeof(double), hipMemcpyDeviceToHost));
	
	// Free the device memory
	hipFree(d_mat_diff);
	hipFree(d_mat_X);
	hipFree(d_mat_a1);
	hipFree(d_mat_W1);
	hipFree(d_mat_W2);
	hipFree(d_mat_dz1);
	hipFree(d_mat_da1);
	hipFree(d_mat_dW1);
	hipFree(d_mat_dW2);
	hipFree(d_mat_db1);
	hipFree(d_mat_db2);
}

/*
 * Do the feedforward and backpropagation in GPU entirely
 * Since this function combines both the feedforward and
 * backpropagation algorithm, some communication cost over
 * the the PCI express is saved such as the cost of transfering
 * the data of sub-matrix X, matrices W1 and W2
 * the second GEMM algorithm is used
 */
void gpu_accel_feedforward_backprop_1 (const double reg,
                                       const double* const mat_X, int X_n_rows, int X_n_cols,
                                       const double* const mat_y, int y_n_rows, int y_n_cols,
                                       const double* const mat_W1, int W1_n_rows, int W1_n_cols,
                                       const double* const mat_b1, int b1_n_rows, int b1_n_cols,
                                       double* mat_z1, int z1_n_rows, int z1_n_cols,
                                       double* mat_a1, int a1_n_rows, int a1_n_cols,
                                       const double* const mat_W2, int W2_n_rows, int W2_n_cols,
                                       const double* const mat_b2, int b2_n_rows, int b2_n_cols,
                                       double* mat_z2, int z2_n_rows, int z2_n_cols,
                                       double* mat_a2, int a2_n_rows, int a2_n_cols,
                                       double* mat_dW1, const int dW1_n_rows, const int dW1_n_cols,
                                       double* mat_dW2, const int dW2_n_rows, const int dW2_n_cols,
                                       double* mat_db1, const int db1_n_cols,
                                       double* mat_db2, const int db2_n_cols)
{
	double* d_mat_X;
	double* d_mat_W1;
	double* d_mat_b1;
	double* d_mat_z1;
	double* d_mat_a1;
	double* d_mat_W2;
	double* d_mat_b2;
	double* d_mat_z2;
	double* d_mat_a2;
	
	double* d_mat_dz1;
	double* d_mat_da1;
	double* d_mat_dW1;
	double* d_mat_dW2;
	double* d_mat_db1;
	double* d_mat_db2;
	
	double* d_mat_y;
	double* d_mat_diff;
	
	dim3 n_threads(0, 0);
	dim3 n_blocks(0, 0);
	
	/*
	 * Allocate the device memory
	 */
	
	checkCudaErrors(hipMalloc(&d_mat_X, X_n_rows*X_n_cols*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_W1, W1_n_rows*W1_n_cols*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_b1, b1_n_rows*b1_n_cols*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_z1, z1_n_rows*z1_n_cols*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_a1, a1_n_rows*a1_n_cols*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_W2, W2_n_rows*W2_n_cols*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_b2, b2_n_rows*b2_n_cols*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_z2, z2_n_rows*z2_n_cols*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_a2, a2_n_rows*a2_n_cols*sizeof(double)));
	
	checkCudaErrors(hipMalloc(&d_mat_dz1, y_n_rows*W2_n_cols*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_da1, y_n_rows*W2_n_cols*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_dW1, dW1_n_rows*dW1_n_cols*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_dW2, dW2_n_rows*dW2_n_cols*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_db1, db1_n_cols*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_db2, db2_n_cols*sizeof(double)));
	
	checkCudaErrors(hipMalloc(&d_mat_y, y_n_rows*y_n_cols*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_diff, y_n_rows*y_n_cols*sizeof(double)));
	
	/*
	 * Copy data from the host memory to the device memory
	 */
	
	checkCudaErrors(hipMemcpy(d_mat_X, mat_X, X_n_rows*X_n_cols*sizeof(double), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_mat_W1, mat_W1, W1_n_rows*W1_n_cols*sizeof(double), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_mat_b1, mat_b1, b1_n_rows*b1_n_cols*sizeof(double), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_mat_W2, mat_W2, W2_n_rows*W2_n_cols*sizeof(double), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_mat_b2, mat_b2, b2_n_rows*b2_n_cols*sizeof(double), hipMemcpyHostToDevice));
	
	checkCudaErrors(hipMemcpy(d_mat_y, mat_y, y_n_rows*y_n_cols*sizeof(double), hipMemcpyHostToDevice));
	
	/*
	 * Do the feedforward for z1
	 */
	
	// Set the size of the sub-block
	n_threads.x = BLOCK_SIZE_2;
	n_threads.y = BLOCK_SIZE_2;
	
	// Assume each dimension of the block is less than 65536
	// and compute the grid size
	n_blocks.x = (z1_n_rows + n_threads.x - 1)/n_threads.x;
	n_blocks.y = (z1_n_cols + n_threads.y - 1)/n_threads.y;
	
	// Launch the kernel to do GEMM
	device_GEMM_2 <BLOCK_SIZE_2> <<<n_blocks, n_threads>>> (1.0,
														    1.0,
														    d_mat_X,
														    d_mat_W1,
														    d_mat_b1,
														    d_mat_z1,
														    X_n_rows,
														    X_n_cols,
														    W1_n_rows,
														    false,
														    true);
		
	/*
	 * Compute the sigmoid of z1
	 */
	
	// Compute the block dimension
	n_threads.x = BLOCK_SIZE_x_SIGMOID;
	n_threads.y = BLOCK_SIZE_y_SIGMOID;
	
	// Compute the grid size
	n_blocks.x = (z1_n_rows + n_threads.x - 1)/n_threads.x;
	n_blocks.y = (z1_n_cols + n_threads.y - 1)/n_threads.y;
	
	// Launch the kernel to apply the sigmoid function
	device_sigmoid <<<n_blocks, n_threads>>> (d_mat_z1, d_mat_a1, z1_n_rows, z1_n_cols);
	
	/*
	 * Do the feedforward for z2
	 */
	
	// Set the size of the sub-block
	n_threads.x = BLOCK_SIZE_2;
	n_threads.y = BLOCK_SIZE_2;
	
	// Assume each dimension of the block is less than 65536
	// and compute the grid size
	n_blocks.x = (z2_n_rows + n_threads.x - 1)/n_threads.x;
	n_blocks.y = (z2_n_cols + n_threads.y - 1)/n_threads.y;
	
	// Launch the kernel to do GEMM
	device_GEMM_2 <BLOCK_SIZE_2> <<<n_blocks, n_threads>>> (1.0,
														    1.0,
														    d_mat_a1,
														    d_mat_W2,
														    d_mat_b2,
														    d_mat_z2,
														    a1_n_rows,
														    a1_n_cols,
														    W2_n_rows,
														    false,
														    true);
	
	/*
	 * Compute the softmax of z2
	 */
	
	double* d_col_vec;
	checkCudaErrors(hipMalloc(&d_col_vec, z2_n_rows*sizeof(double)));
	
	// Compute the block dimension
	n_threads.x = BLOCK_SIZE_x_SOFTMAX;
	n_threads.y = BLOCK_SIZE_y_SOFTMAX;
	
	// Compute the grid size
	n_blocks.x = (z2_n_rows + n_threads.x - 1)/n_threads.x;
	n_blocks.y = (z2_n_cols + n_threads.y - 1)/n_threads.y;
	
	// Launch the kernel to compute the elementwise exponentinal function
	device_exponent <<<n_blocks, n_threads>>> (d_mat_z2, d_mat_a2, z2_n_rows, z2_n_cols);
	
	// Compute the block dimension
	n_threads.x = 1;
	n_threads.y = BLOCK_SIZE_REDUCTION;
	
	// Compute the grid size
	n_blocks.x = a2_n_rows;
	n_blocks.y = 1;
	
	device_sum_row_block <BLOCK_SIZE_REDUCTION> <<<n_blocks, n_threads>>> (d_mat_a2, d_col_vec, a2_n_rows, a2_n_cols);
	
	// Compute the block dimension
	n_threads.x = BLOCK_SIZE_x_SOFTMAX;
	n_threads.y = BLOCK_SIZE_y_SOFTMAX;
	
	// Compute the grid size
	n_blocks.x = (a2_n_rows + n_threads.x - 1)/n_threads.x;
	n_blocks.y = (a2_n_cols + n_threads.y - 1)/n_threads.y;
	
	device_normalize <<<n_blocks, n_threads>>> (d_mat_a2, d_col_vec, a2_n_rows, a2_n_cols);
	
	hipFree(d_col_vec);
	
	/*
	 * Compute d_mat_diff
	 */
	
	// Compute the block dimension
	n_threads.x = BLOCK_SIZE_x_DIFF;
	n_threads.y = BLOCK_SIZE_y_DIFF;
	
	// Compute the grid size
	n_blocks.x = (y_n_rows + n_threads.x - 1)/n_threads.x;
	n_blocks.y = (y_n_cols + n_threads.y - 1)/n_threads.y;
	
	// Launch the kernel to apply the sigmoid function
	device_compute_diff <<<n_blocks, n_threads>>> (d_mat_a2, d_mat_y, d_mat_diff, y_n_rows, y_n_cols);
	
	/*
	 * Compute dW2
	 */
	
	// Set the size of the sub-block
	n_threads.x = BLOCK_SIZE_2;
	n_threads.y = BLOCK_SIZE_2;
	
	// Assume each dimension of the block is less than 65536
	// and compute the grid size
	n_blocks.x = (dW2_n_rows + n_threads.x - 1)/n_threads.x;
	n_blocks.y = (dW2_n_cols + n_threads.y - 1)/n_threads.y;
	
	// Launch the kernel to do GEMM
	device_GEMM_2 <BLOCK_SIZE_2> <<<n_blocks, n_threads>>> (1.0,
														    reg,
														    d_mat_diff,
														    d_mat_a1,
														    d_mat_W2,
														    d_mat_dW2,
														    y_n_cols,
														    y_n_rows,
														    a1_n_cols,
														    true,
														    false);
	
	/*
	 * Compute dW1
	 */
	
	n_threads.x = BLOCK_SIZE_2;
	n_threads.y = BLOCK_SIZE_2;
	
	// Assume each dimension of the block is less than 65536
	// and compute the grid size
	n_blocks.x = (y_n_rows + n_threads.x - 1)/n_threads.x;
	n_blocks.y = (W2_n_cols + n_threads.y - 1)/n_threads.y;
	
	// Launch the kernel to do GEMM
	device_GEMM_2 <BLOCK_SIZE_2> <<<n_blocks, n_threads>>> (1.0,
														    0.0,
														    d_mat_diff,
														    d_mat_W2,
														    d_mat_da1,
														    d_mat_da1,
														    y_n_rows,
														    y_n_cols,
														    W2_n_cols,
														    false,
														    false);

	// Compute the block dimension
	n_threads.x = BLOCK_SIZE_x_elementwise_mult;
	n_threads.y = BLOCK_SIZE_y_elementwise_mult;
	
	// Compute the grid size
	n_blocks.x = (y_n_rows + n_threads.x - 1)/n_threads.x;
	n_blocks.y = (W2_n_cols + n_threads.y - 1)/n_threads.y;
	
	// Launch the kernel to do the elementwise multiplication
	device_elementwise_mult <<<n_blocks, n_threads>>> (d_mat_da1, d_mat_a1, d_mat_dz1, y_n_rows, W2_n_cols);
	
	n_threads.x = BLOCK_SIZE_2;
	n_threads.y = BLOCK_SIZE_2;
	
	// Assume each dimension of the block is less than 65536
	// and compute the grid size
	n_blocks.x = (W2_n_cols + n_threads.x - 1)/n_threads.x;
	n_blocks.y = (X_n_cols + n_threads.y - 1)/n_threads.y;
	
	// Launch the kernel to do GEMM
	device_GEMM_2 <BLOCK_SIZE_2> <<<n_blocks, n_threads>>> (1.0,
														    reg,
														    d_mat_dz1,
														    d_mat_X,
														    d_mat_W1,
														    d_mat_dW1,
														    W2_n_cols,
														    y_n_rows,
														    X_n_cols,
														    true,
														    false);
	
	/*
	 * Compute db1
	 */
	
	// Compute the block dimension
	n_threads.x = BLOCK_SIZE_REDUCTION;
	n_threads.y = 1;
	
	// Compute the grid size
	n_blocks.x = 1;
	n_blocks.y = y_n_rows;
	
	device_sum_col_block <BLOCK_SIZE_REDUCTION> <<<n_blocks, n_threads>>> (d_mat_diff, d_mat_db2, y_n_cols, y_n_rows);
	
	/*
	 * Compute db2
	 */
	
	// Compute the block dimension
	n_threads.x = BLOCK_SIZE_REDUCTION;
	n_threads.y = 1;
	
	// Compute the grid size
	n_blocks.x = 1;
	n_blocks.y = W2_n_cols;
	
	device_sum_col_block <BLOCK_SIZE_REDUCTION> <<<n_blocks, n_threads>>> (d_mat_dz1, d_mat_db1, y_n_rows, W2_n_cols);
	
	/*
	 * Copy data from the device memory to the host memory
	 */
	
	checkCudaErrors(hipMemcpy(mat_z1, d_mat_z1, z1_n_rows*z1_n_cols*sizeof(double), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(mat_a1, d_mat_a1, a1_n_rows*a1_n_cols*sizeof(double), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(mat_z2, d_mat_z2, z2_n_rows*z2_n_cols*sizeof(double), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(mat_a2, d_mat_a2, a2_n_rows*a2_n_cols*sizeof(double), hipMemcpyDeviceToHost));
	
	checkCudaErrors(hipMemcpy(mat_dW1, d_mat_dW1, dW1_n_rows*dW1_n_cols*sizeof(double), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(mat_dW2, d_mat_dW2, dW2_n_rows*dW2_n_cols*sizeof(double), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(mat_db1, d_mat_db1, db1_n_cols*sizeof(double), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(mat_db2, d_mat_db2, db2_n_cols*sizeof(double), hipMemcpyDeviceToHost));
		
	/*
	 * Free the device memory
	 */
	
	hipFree(d_mat_X);
	hipFree(d_mat_W1);
	hipFree(d_mat_b1);
	hipFree(d_mat_z1);
	hipFree(d_mat_a1);
	hipFree(d_mat_W2);
	hipFree(d_mat_b2);
	hipFree(d_mat_z2);
	hipFree(d_mat_a2);
	
	hipFree(d_mat_dz1);
	hipFree(d_mat_da1);
	hipFree(d_mat_dW1);
	hipFree(d_mat_dW2);
	hipFree(d_mat_db1);
	hipFree(d_mat_db2);
	
	hipFree(d_mat_y);
	hipFree(d_mat_diff);
}

/*
 * The stream version of gpu_accel_feedforward_backprop_2
 * This function is not preferred since the cost of pinning
 * host memory is large
 */
void gpu_accel_feedforward_backprop_2_w_stream (const double reg,
											    double* mat_X, int X_n_rows, int X_n_cols,
												double* mat_y, int y_n_rows, int y_n_cols,
												double* mat_W1, int W1_n_rows, int W1_n_cols,
												double* mat_b1, int b1_n_rows, int b1_n_cols,
												double* mat_W2, int W2_n_rows, int W2_n_cols,
												double* mat_b2, int b2_n_rows, int b2_n_cols,
												double* mat_a2, int a2_n_rows, int a2_n_cols,
												double* mat_dW1, const int dW1_n_rows, const int dW1_n_cols,
												double* mat_dW2, const int dW2_n_rows, const int dW2_n_cols,
												double* mat_db1, const int db1_n_cols,
												double* mat_db2, const int db2_n_cols)
{
	double* d_mat_X;
	double* d_mat_W1;
	double* d_mat_b1;
	double* d_mat_z1;
	double* d_mat_W2;
	double* d_mat_b2;
	double* d_mat_z2;
	
	double* d_mat_dz1;
	double* d_mat_da1;
	double* d_mat_dW1;
	double* d_mat_dW2;
	double* d_mat_db1;
	double* d_mat_db2;
	
	double* d_mat_y;
	double* d_mat_diff;
	
	dim3 n_threads(0, 0);
	dim3 n_blocks(0, 0);
	
	/*
	 * Create user streams
	 */
	hipStream_t stream1, stream2;
	hipStreamCreate(&stream1);
	hipStreamCreate(&stream2);
	
	/*
	 * Allocate the device memory
	 */
	
	checkCudaErrors(hipMalloc(&d_mat_X, X_n_rows*X_n_cols*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_W1, W1_n_rows*W1_n_cols*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_b1, b1_n_rows*b1_n_cols*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_z1, X_n_rows*W1_n_rows*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_W2, W2_n_rows*W2_n_cols*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_b2, b2_n_rows*b2_n_cols*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_z2, X_n_rows*W2_n_rows*sizeof(double)));
	
	checkCudaErrors(hipMalloc(&d_mat_dz1, y_n_rows*W2_n_cols*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_da1, y_n_rows*W2_n_cols*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_dW1, dW1_n_rows*dW1_n_cols*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_dW2, dW2_n_rows*dW2_n_cols*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_db1, db1_n_cols*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_db2, db2_n_cols*sizeof(double)));
	
	checkCudaErrors(hipMalloc(&d_mat_y, y_n_rows*y_n_cols*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_diff, y_n_rows*y_n_cols*sizeof(double)));
	
	/*
	 * Pin the host memory
	 */
	
	checkCudaErrors(hipHostRegister(mat_X, X_n_rows*X_n_cols*sizeof(double), 0));
	checkCudaErrors(hipHostRegister(mat_W1, W1_n_rows*W1_n_cols*sizeof(double), 0));
	checkCudaErrors(hipHostRegister(mat_b1, b1_n_rows*b1_n_cols*sizeof(double), 0));
	checkCudaErrors(hipHostRegister(mat_W2, W2_n_rows*W2_n_cols*sizeof(double), 0));
	checkCudaErrors(hipHostRegister(mat_b2, b2_n_rows*b2_n_cols*sizeof(double), 0));
	checkCudaErrors(hipHostRegister(mat_y, y_n_rows*y_n_cols*sizeof(double), 0));
	
	checkCudaErrors(hipHostRegister(mat_a2, a2_n_rows*a2_n_cols*sizeof(double), 0));
	
	checkCudaErrors(hipHostRegister(mat_dW1, dW1_n_rows*dW1_n_cols*sizeof(double), 0));
	checkCudaErrors(hipHostRegister(mat_dW2, dW2_n_rows*dW2_n_cols*sizeof(double), 0));
	checkCudaErrors(hipHostRegister(mat_db1, db1_n_cols*sizeof(double), 0));
	checkCudaErrors(hipHostRegister(mat_db2, db2_n_cols*sizeof(double), 0));
	
	/*
	 * Copy data from the host memory to the device memory
	 */
	
	checkCudaErrors(hipMemcpyAsync(d_mat_X, mat_X, X_n_rows*X_n_cols*sizeof(double), hipMemcpyHostToDevice, stream1));
	checkCudaErrors(hipMemcpyAsync(d_mat_W1, mat_W1, W1_n_rows*W1_n_cols*sizeof(double), hipMemcpyHostToDevice, stream1));
	checkCudaErrors(hipMemcpyAsync(d_mat_b1, mat_b1, b1_n_rows*b1_n_cols*sizeof(double), hipMemcpyHostToDevice, stream1));
	
	checkCudaErrors(hipMemcpyAsync(d_mat_W2, mat_W2, W2_n_rows*W2_n_cols*sizeof(double), hipMemcpyHostToDevice, stream2));
	checkCudaErrors(hipMemcpyAsync(d_mat_b2, mat_b2, b2_n_rows*b2_n_cols*sizeof(double), hipMemcpyHostToDevice, stream2));
	
	/*
	 * Do the feedforward for z1
	 */
	
	// Set the size of the sub-block
	n_threads.x = BLOCK_SIZE_2;
	n_threads.y = BLOCK_SIZE_2;
	
	// Assume each dimension of the block is less than 65536
	// and compute the grid size
	n_blocks.x = (X_n_rows + n_threads.x - 1)/n_threads.x;
	n_blocks.y = (W1_n_rows + n_threads.y - 1)/n_threads.y;
	
	// Launch the kernel to do GEMM
	device_GEMM_2 <BLOCK_SIZE_2> <<<n_blocks, n_threads, 0, stream1>>> (1.0,
														    1.0,
														    d_mat_X,
														    d_mat_W1,
														    d_mat_b1,
														    d_mat_z1,
														    X_n_rows,
														    X_n_cols,
														    W1_n_rows,
														    false,
														    true);
		
	/*
	 * Compute the sigmoid of z1
	 */
	
	// Compute the block dimension
	n_threads.x = BLOCK_SIZE_x_SIGMOID;
	n_threads.y = BLOCK_SIZE_y_SIGMOID;
	
	// Compute the grid size
	n_blocks.x = (X_n_rows + n_threads.x - 1)/n_threads.x;
	n_blocks.y = (W1_n_rows + n_threads.y - 1)/n_threads.y;
	
	// Launch the kernel to apply the sigmoid function
	device_sigmoid <<<n_blocks, n_threads, 0, stream1>>> (d_mat_z1, d_mat_z1, X_n_rows, W1_n_rows);
	
	// Synchronize to make sure d_mat_W2, d_mat_b2 are already loaded
	hipDeviceSynchronize();
	checkCudaErrors(hipMemcpyAsync(d_mat_y, mat_y, y_n_rows*y_n_cols*sizeof(double), hipMemcpyHostToDevice, stream2));
	
	/*
	 * Do the feedforward for z2
	 */
	
	// Set the size of the sub-block
	n_threads.x = BLOCK_SIZE_2;
	n_threads.y = BLOCK_SIZE_2;
	
	// Assume each dimension of the block is less than 65536
	// and compute the grid size
	n_blocks.x = (X_n_rows + n_threads.x - 1)/n_threads.x;
	n_blocks.y = (W2_n_rows + n_threads.y - 1)/n_threads.y;
	
	// Launch the kernel to do GEMM
	device_GEMM_2 <BLOCK_SIZE_2> <<<n_blocks, n_threads, 0, stream1>>> (1.0,
														    1.0,
														    d_mat_z1,
														    d_mat_W2,
														    d_mat_b2,
														    d_mat_z2,
														    X_n_rows,
														    W1_n_rows,
														    W2_n_rows,
														    false,
														    true);
	
	/*
	 * Compute the softmax of z2
	 */
	
	double* d_col_vec;
	checkCudaErrors(hipMalloc(&d_col_vec, X_n_rows*sizeof(double)));
	
	// Compute the block dimension
	n_threads.x = BLOCK_SIZE_x_SOFTMAX;
	n_threads.y = BLOCK_SIZE_y_SOFTMAX;
	
	// Compute the grid size
	n_blocks.x = (X_n_rows + n_threads.x - 1)/n_threads.x;
	n_blocks.y = (W2_n_rows + n_threads.y - 1)/n_threads.y;
	
	// Launch the kernel to compute the elementwise exponentinal function
	device_exponent <<<n_blocks, n_threads, 0, stream1>>> (d_mat_z2, d_mat_z2, X_n_rows, W2_n_rows);
	
	// Compute the block dimension
	n_threads.x = 1;
	n_threads.y = BLOCK_SIZE_REDUCTION;
	
	// Compute the grid size
	n_blocks.x = a2_n_rows;
	n_blocks.y = 1;
	
	device_sum_row_block <BLOCK_SIZE_REDUCTION> <<<n_blocks, n_threads, 0, stream1>>> (d_mat_z2, d_col_vec, a2_n_rows, a2_n_cols);
	
	// Compute the block dimension
	n_threads.x = BLOCK_SIZE_x_SOFTMAX;
	n_threads.y = BLOCK_SIZE_y_SOFTMAX;
	
	// Compute the grid size
	n_blocks.x = (a2_n_rows + n_threads.x - 1)/n_threads.x;
	n_blocks.y = (a2_n_cols + n_threads.y - 1)/n_threads.y;
	
	device_normalize <<<n_blocks, n_threads, 0, stream1>>> (d_mat_z2, d_col_vec, a2_n_rows, a2_n_cols);
	
	hipFree(d_col_vec);
	
	// Synchronize to make sure d_mat_y is already loaded
	hipDeviceSynchronize();
	
	/*
	 * Compute d_mat_diff
	 */
	
	// Compute the block dimension
	n_threads.x = BLOCK_SIZE_x_DIFF;
	n_threads.y = BLOCK_SIZE_y_DIFF;
	
	// Compute the grid size
	n_blocks.x = (y_n_rows + n_threads.x - 1)/n_threads.x;
	n_blocks.y = (y_n_cols + n_threads.y - 1)/n_threads.y;
	
	// Launch the kernel to apply the sigmoid function
	device_compute_diff <<<n_blocks, n_threads, 0, stream1>>> (d_mat_z2, d_mat_y, d_mat_diff, y_n_rows, y_n_cols);
	
	/*
	 * Compute dW2
	 */
	
	// Set the size of the sub-block
	n_threads.x = BLOCK_SIZE_2;
	n_threads.y = BLOCK_SIZE_2;
	
	// Assume each dimension of the block is less than 65536
	// and compute the grid size
	n_blocks.x = (dW2_n_rows + n_threads.x - 1)/n_threads.x;
	n_blocks.y = (dW2_n_cols + n_threads.y - 1)/n_threads.y;
	
	// Launch the kernel to do GEMM
	device_GEMM_2 <BLOCK_SIZE_2> <<<n_blocks, n_threads, 0, stream1>>> (1.0,
														    reg,
														    d_mat_diff,
														    d_mat_z1,
														    d_mat_W2,
														    d_mat_dW2,
														    y_n_cols,
														    y_n_rows,
														    W1_n_rows,
														    true,
														    false);
	
	/*
	 * Compute dW1
	 */
	
	n_threads.x = BLOCK_SIZE_2;
	n_threads.y = BLOCK_SIZE_2;
	
	// Assume each dimension of the block is less than 65536
	// and compute the grid size
	n_blocks.x = (y_n_rows + n_threads.x - 1)/n_threads.x;
	n_blocks.y = (W2_n_cols + n_threads.y - 1)/n_threads.y;
	
	// Launch the kernel to do GEMM
	device_GEMM_2 <BLOCK_SIZE_2> <<<n_blocks, n_threads, 0, stream1>>> (1.0,
														    0.0,
														    d_mat_diff,
														    d_mat_W2,
														    d_mat_da1,
														    d_mat_da1,
														    y_n_rows,
														    y_n_cols,
														    W2_n_cols,
														    false,
														    false);

	// Compute the block dimension
	n_threads.x = BLOCK_SIZE_x_elementwise_mult;
	n_threads.y = BLOCK_SIZE_y_elementwise_mult;
	
	// Compute the grid size
	n_blocks.x = (y_n_rows + n_threads.x - 1)/n_threads.x;
	n_blocks.y = (W2_n_cols + n_threads.y - 1)/n_threads.y;
	
	// Launch the kernel to do the elementwise multiplication
	device_elementwise_mult <<<n_blocks, n_threads, 0, stream1>>> (d_mat_da1, d_mat_z1, d_mat_dz1, y_n_rows, W2_n_cols);
	
	n_threads.x = BLOCK_SIZE_2;
	n_threads.y = BLOCK_SIZE_2;
	
	// Assume each dimension of the block is less than 65536
	// and compute the grid size
	n_blocks.x = (W2_n_cols + n_threads.x - 1)/n_threads.x;
	n_blocks.y = (X_n_cols + n_threads.y - 1)/n_threads.y;
	
	// Launch the kernel to do GEMM
	device_GEMM_2 <BLOCK_SIZE_2> <<<n_blocks, n_threads, 0, stream1>>> (1.0,
														    reg,
														    d_mat_dz1,
														    d_mat_X,
														    d_mat_W1,
														    d_mat_dW1,
														    W2_n_cols,
														    y_n_rows,
														    X_n_cols,
														    true,
														    false);
	
	/*
	 * Compute db1
	 */
	
	// Compute the block dimension
	n_threads.x = BLOCK_SIZE_REDUCTION;
	n_threads.y = 1;
	
	// Compute the grid size
	n_blocks.x = 1;
	n_blocks.y = y_n_rows;
	
	device_sum_col_block <BLOCK_SIZE_REDUCTION> <<<n_blocks, n_threads, 0, stream1>>> (d_mat_diff, d_mat_db2, y_n_cols, y_n_rows);
	
	/*
	 * Compute db2
	 */
	
	// Compute the block dimension
	n_threads.x = BLOCK_SIZE_REDUCTION;
	n_threads.y = 1;
	
	// Compute the grid size
	n_blocks.x = 1;
	n_blocks.y = W2_n_cols;
	
	device_sum_col_block <BLOCK_SIZE_REDUCTION> <<<n_blocks, n_threads, 0, stream1>>> (d_mat_dz1, d_mat_db1, y_n_rows, W2_n_cols);
	
	/*
	 * Copy data from the device memory to the host memory
	 */
	
	hipDeviceSynchronize();
	
	checkCudaErrors(hipMemcpyAsync(mat_dW1, d_mat_dW1, dW1_n_rows*dW1_n_cols*sizeof(double), hipMemcpyDeviceToHost, stream2));
	checkCudaErrors(hipMemcpyAsync(mat_a2, d_mat_z2, a2_n_rows*a2_n_cols*sizeof(double), hipMemcpyDeviceToHost, stream2));
	checkCudaErrors(hipMemcpyAsync(mat_db1, d_mat_db1, db1_n_cols*sizeof(double), hipMemcpyDeviceToHost, stream2));
	checkCudaErrors(hipMemcpyAsync(mat_db2, d_mat_db2, db2_n_cols*sizeof(double), hipMemcpyDeviceToHost, stream2));
	checkCudaErrors(hipMemcpyAsync(mat_dW2, d_mat_dW2, dW2_n_rows*dW2_n_cols*sizeof(double), hipMemcpyDeviceToHost, stream2));
		
	/*
	 * Free the device memory
	 */
	
	hipFree(d_mat_X);
	hipFree(d_mat_W1);
	hipFree(d_mat_b1);
	hipFree(d_mat_z1);
	hipFree(d_mat_W2);
	hipFree(d_mat_b2);
	hipFree(d_mat_z2);
	
	hipFree(d_mat_dz1);
	hipFree(d_mat_da1);
	hipFree(d_mat_dW1);
	hipFree(d_mat_dW2);
	hipFree(d_mat_db1);
	hipFree(d_mat_db2);
	
	hipFree(d_mat_y);
	hipFree(d_mat_diff);
	
	/*
	 * Unpin the host memory
	 */
	
	checkCudaErrors(hipHostUnregister(mat_X));
	checkCudaErrors(hipHostUnregister(mat_W1));
	checkCudaErrors(hipHostUnregister(mat_b1));
	checkCudaErrors(hipHostUnregister(mat_W2));
	checkCudaErrors(hipHostUnregister(mat_b2));
	checkCudaErrors(hipHostUnregister(mat_y));
	
	checkCudaErrors(hipHostUnregister(mat_a2));
	
	checkCudaErrors(hipHostUnregister(mat_dW1));
	checkCudaErrors(hipHostUnregister(mat_dW2));
	checkCudaErrors(hipHostUnregister(mat_db1));
	checkCudaErrors(hipHostUnregister(mat_db2));
	
	/*
	 * Destroy user streams
	 */
	hipStreamDestroy(stream1);
	hipStreamDestroy(stream2);	
}

/*
 * Do the feedforward and backpropagation in GPU entirely
 * Compared to gpu_accel_feedforward_backprop_1, this function
 * further minimizes the communication cost. Redundant communication
 * such as transferring back data of z1, a1, z2 from GPU
 * Also, the third GEMM algorithm, which is faster, is used
 */
void gpu_accel_feedforward_backprop_2 (const double reg,
                                       double* mat_X, int X_n_rows, int X_n_cols,
                                       double* mat_y, int y_n_rows, int y_n_cols,
                                       double* mat_W1, int W1_n_rows, int W1_n_cols,
                                       double* mat_b1, int b1_n_rows, int b1_n_cols,
                                       double* mat_W2, int W2_n_rows, int W2_n_cols,
                                       double* mat_b2, int b2_n_rows, int b2_n_cols,
                                       double* mat_a2, int a2_n_rows, int a2_n_cols,
                                       double* mat_dW1, const int dW1_n_rows, const int dW1_n_cols,
                                       double* mat_dW2, const int dW2_n_rows, const int dW2_n_cols,
                                       double* mat_db1, const int db1_n_cols,
                                       double* mat_db2, const int db2_n_cols)
{
	double* d_mat_X;
	double* d_mat_W1;
	double* d_mat_b1;
	double* d_mat_z1;
	double* d_mat_W2;
	double* d_mat_b2;
	double* d_mat_z2;
	
	double* d_mat_dz1;
	double* d_mat_da1;
	double* d_mat_dW1;
	double* d_mat_dW2;
	double* d_mat_db1;
	double* d_mat_db2;
	
	double* d_mat_y;
	double* d_mat_diff;
	
	double* d_mat_W1_t;
	double* d_mat_W2_t;
	double* d_mat_diff_t;
	double* d_mat_dz1_t;
	
	dim3 n_threads(0, 0);
	dim3 n_blocks(0, 0);

	/*
	 * Allocate the device memory
	 */
	
	checkCudaErrors(hipMalloc(&d_mat_X, X_n_rows*X_n_cols*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_W1, W1_n_rows*W1_n_cols*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_b1, b1_n_rows*b1_n_cols*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_z1, X_n_rows*W1_n_rows*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_W2, W2_n_rows*W2_n_cols*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_b2, b2_n_rows*b2_n_cols*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_z2, X_n_rows*W2_n_rows*sizeof(double)));
	
	checkCudaErrors(hipMalloc(&d_mat_dz1, y_n_rows*W2_n_cols*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_da1, y_n_rows*W2_n_cols*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_dW1, dW1_n_rows*dW1_n_cols*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_dW2, dW2_n_rows*dW2_n_cols*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_db1, db1_n_cols*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_db2, db2_n_cols*sizeof(double)));
	
	checkCudaErrors(hipMalloc(&d_mat_y, y_n_rows*y_n_cols*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_diff, y_n_rows*y_n_cols*sizeof(double)));
	
	checkCudaErrors(hipMalloc(&d_mat_W1_t, W1_n_rows*W1_n_cols*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_W2_t, W2_n_rows*W2_n_cols*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_diff_t, y_n_rows*y_n_cols*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_mat_dz1_t, y_n_rows*W2_n_cols*sizeof(double)));
	
	/*
	 * Copy data from the host memory to the device memory
	 */
	
	checkCudaErrors(hipMemcpy(d_mat_X, mat_X, X_n_rows*X_n_cols*sizeof(double), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_mat_W1, mat_W1, W1_n_rows*W1_n_cols*sizeof(double), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_mat_b1, mat_b1, b1_n_rows*b1_n_cols*sizeof(double), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_mat_W2, mat_W2, W2_n_rows*W2_n_cols*sizeof(double), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_mat_b2, mat_b2, b2_n_rows*b2_n_cols*sizeof(double), hipMemcpyHostToDevice));
	
	checkCudaErrors(hipMemcpy(d_mat_y, mat_y, y_n_rows*y_n_cols*sizeof(double), hipMemcpyHostToDevice));
	
	/*
	 * Do the feedforward for z1
	 */
	
	// Transpose d_mat_W1
	// Compute the block dimension
	n_threads.x = BLOCK_SIZE_x_transpose;
	n_threads.y = BLOCK_SIZE_y_transpose;
	
	// Compute the grid size
	n_blocks.x = (W1_n_rows + n_threads.x - 1)/n_threads.x;
	n_blocks.y = (W1_n_cols + n_threads.y - 1)/n_threads.y;
	
	// Launch the kernel to do the transpose
	device_transpose <<<n_blocks, n_threads>>> (d_mat_W1, d_mat_W1_t, W1_n_rows, W1_n_cols);
	
	// Set the size of each block
	n_threads.x = BLOCK_SIZE_x_3;
	n_threads.y = BLOCK_SIZE_y_3;
	
	// Assume each dimension of the block is less than 65536
	// and compute the grid size
	n_blocks.x = (X_n_rows + n_threads.x*n_threads.y - 1)/(n_threads.x*n_threads.y);
	n_blocks.y = (W1_n_rows + n_threads.x - 1)/n_threads.x;
	
	// Launch the kernel to do GEMM
	device_GEMM_3 <BLOCK_SIZE_x_3, BLOCK_SIZE_y_3> <<<n_blocks, n_threads>>> (1.0,
																			  1.0,
																			  d_mat_X,
																			  d_mat_W1_t,
																			  d_mat_b1,
																			  d_mat_z1,
																			  X_n_rows,
																			  X_n_cols,
																			  W1_n_rows,
																			  false,
																			  false);
	
	/*
	 * Compute the sigmoid of z1
	 */
	
	// Compute the block dimension
	n_threads.x = BLOCK_SIZE_x_SIGMOID;
	n_threads.y = BLOCK_SIZE_y_SIGMOID;
	
	// Compute the grid size
	n_blocks.x = (X_n_rows + n_threads.x - 1)/n_threads.x;
	n_blocks.y = (W1_n_rows + n_threads.y - 1)/n_threads.y;
	
	// Launch the kernel to apply the sigmoid function
	device_sigmoid <<<n_blocks, n_threads>>> (d_mat_z1, d_mat_z1, X_n_rows, W1_n_rows);
	
	/*
	 * Do the feedforward for z2
	 */
	
	// Transpose d_mat_W2
	// Compute the block dimension
	n_threads.x = BLOCK_SIZE_x_transpose;
	n_threads.y = BLOCK_SIZE_y_transpose;
	
	// Compute the grid size
	n_blocks.x = (W2_n_rows + n_threads.x - 1)/n_threads.x;
	n_blocks.y = (W2_n_cols + n_threads.y - 1)/n_threads.y;
	
	// Launch the kernel to do the transpose
	device_transpose <<<n_blocks, n_threads>>> (d_mat_W2, d_mat_W2_t, W2_n_rows, W2_n_cols);
	
	// Set the size of each block
	n_threads.x = BLOCK_SIZE_x_3;
	n_threads.y = BLOCK_SIZE_y_3;
	
	// Assume each dimension of the block is less than 65536
	// and compute the grid size
	n_blocks.x = (X_n_rows + n_threads.x*n_threads.y - 1)/(n_threads.x*n_threads.y);
	n_blocks.y = (W2_n_rows + n_threads.x - 1)/n_threads.x;
	
	// Launch the kernel to do GEMM
	device_GEMM_3 <BLOCK_SIZE_x_3, BLOCK_SIZE_y_3> <<<n_blocks, n_threads>>> (1.0,
																			  1.0,
																			  d_mat_z1,
																			  d_mat_W2_t,
																			  d_mat_b2,
																			  d_mat_z2,
																			  X_n_rows,
																			  W1_n_rows,
																			  W2_n_rows,
																			  false,
																			  false);

	/*
	 * Compute the softmax of z2
	 */
	
	double* d_col_vec;
	checkCudaErrors(hipMalloc(&d_col_vec, X_n_rows*sizeof(double)));
	
	// Compute the block dimension
	n_threads.x = BLOCK_SIZE_x_SOFTMAX;
	n_threads.y = BLOCK_SIZE_y_SOFTMAX;
	
	// Compute the grid size
	n_blocks.x = (X_n_rows + n_threads.x - 1)/n_threads.x;
	n_blocks.y = (W2_n_rows + n_threads.y - 1)/n_threads.y;
	
	// Launch the kernel to compute the elementwise exponentinal function
	device_exponent <<<n_blocks, n_threads>>> (d_mat_z2, d_mat_z2, X_n_rows, W2_n_rows);
	
	// Compute the block dimension
	n_threads.x = 1;
	n_threads.y = BLOCK_SIZE_REDUCTION;
	
	// Compute the grid size
	n_blocks.x = a2_n_rows;
	n_blocks.y = 1;
	
	device_sum_row_block <BLOCK_SIZE_REDUCTION> <<<n_blocks, n_threads>>> (d_mat_z2, d_col_vec, a2_n_rows, a2_n_cols);
	
	// Compute the block dimension
	n_threads.x = BLOCK_SIZE_x_SOFTMAX;
	n_threads.y = BLOCK_SIZE_y_SOFTMAX;
	
	// Compute the grid size
	n_blocks.x = (a2_n_rows + n_threads.x - 1)/n_threads.x;
	n_blocks.y = (a2_n_cols + n_threads.y - 1)/n_threads.y;
	
	device_normalize <<<n_blocks, n_threads>>> (d_mat_z2, d_col_vec, a2_n_rows, a2_n_cols);
	
	hipFree(d_col_vec);

	/*
	 * Compute d_mat_diff
	 */
	
	// Compute the block dimension
	n_threads.x = BLOCK_SIZE_x_DIFF;
	n_threads.y = BLOCK_SIZE_y_DIFF;
	
	// Compute the grid size
	n_blocks.x = (y_n_rows + n_threads.x - 1)/n_threads.x;
	n_blocks.y = (y_n_cols + n_threads.y - 1)/n_threads.y;
	
	// Launch the kernel to apply the sigmoid function
	device_compute_diff <<<n_blocks, n_threads>>> (d_mat_z2, d_mat_y, d_mat_diff, y_n_rows, y_n_cols);

	/*
	 * Compute dW2
	 */
	
	// Transpose d_mat_diff
	// Compute the block dimension
	n_threads.x = BLOCK_SIZE_x_transpose;
	n_threads.y = BLOCK_SIZE_y_transpose;
	
	// Compute the grid size
	n_blocks.x = (y_n_rows + n_threads.x - 1)/n_threads.x;
	n_blocks.y = (y_n_cols + n_threads.y - 1)/n_threads.y;
	
	// Launch the kernel to do the transpose
	device_transpose <<<n_blocks, n_threads>>> (d_mat_diff, d_mat_diff_t, y_n_rows, y_n_cols);
	
	// Set the size of each block
	n_threads.x = BLOCK_SIZE_x_3;
	n_threads.y = BLOCK_SIZE_y_3;
	
	// Assume each dimension of the block is less than 65536
	// and compute the grid size
	n_blocks.x = (dW2_n_rows + n_threads.x*n_threads.y - 1)/(n_threads.x*n_threads.y);
	n_blocks.y = (dW2_n_cols + n_threads.x - 1)/n_threads.x;
	
	// Launch the kernel to do GEMM
	device_GEMM_3 <BLOCK_SIZE_x_3, BLOCK_SIZE_y_3> <<<n_blocks, n_threads>>> (1.0,
																			  reg,
																			  d_mat_diff_t,
																			  d_mat_z1,
																			  d_mat_W2,
																			  d_mat_dW2,
																			  y_n_cols,
																			  y_n_rows,
																			  W1_n_rows,
																			  false,
																			  false);

	/*
	 * Compute dW1
	 */
	
	// Set the size of each block
	n_threads.x = BLOCK_SIZE_x_3;
	n_threads.y = BLOCK_SIZE_y_3;
	
	// Assume each dimension of the block is less than 65536
	// and compute the grid size
	n_blocks.x = (y_n_rows + n_threads.x*n_threads.y - 1)/(n_threads.x*n_threads.y);
	n_blocks.y = (W2_n_cols + n_threads.x - 1)/n_threads.x;
	
	// Launch the kernel to do GEMM
	device_GEMM_3 <BLOCK_SIZE_x_3, BLOCK_SIZE_y_3> <<<n_blocks, n_threads>>> (1.0,
																			  0.0,
																			  d_mat_diff,
																			  d_mat_W2,
																			  d_mat_da1,
																			  d_mat_da1,
																			  y_n_rows,
																			  y_n_cols,
																			  W2_n_cols,
																			  false,
																			  false);

	// Compute the block dimension
	n_threads.x = BLOCK_SIZE_x_elementwise_mult;
	n_threads.y = BLOCK_SIZE_y_elementwise_mult;
	
	// Compute the grid size
	n_blocks.x = (y_n_rows + n_threads.x - 1)/n_threads.x;
	n_blocks.y = (W2_n_cols + n_threads.y - 1)/n_threads.y;
	
	// Launch the kernel to do the elementwise multiplication
	device_elementwise_mult <<<n_blocks, n_threads>>> (d_mat_da1, d_mat_z1, d_mat_dz1, y_n_rows, W2_n_cols);
	
	// Transpose d_mat_dz1
	// Compute the block dimension
	n_threads.x = BLOCK_SIZE_x_transpose;
	n_threads.y = BLOCK_SIZE_y_transpose;
	
	// Compute the grid size
	n_blocks.x = (y_n_rows + n_threads.x - 1)/n_threads.x;
	n_blocks.y = (W2_n_cols + n_threads.y - 1)/n_threads.y;
	
	// Launch the kernel to do the transpose
	device_transpose <<<n_blocks, n_threads>>> (d_mat_dz1, d_mat_dz1_t, y_n_rows, W2_n_cols);
	
	// Set the size of each block
	n_threads.x = BLOCK_SIZE_x_3;
	n_threads.y = BLOCK_SIZE_y_3;
	
	// Assume each dimension of the block is less than 65536
	// and compute the grid size
	n_blocks.x = (W2_n_cols + n_threads.x*n_threads.y - 1)/(n_threads.x*n_threads.y);
	n_blocks.y = (X_n_cols + n_threads.x - 1)/n_threads.x;
	
	// Launch the kernel to do GEMM
	device_GEMM_3 <BLOCK_SIZE_x_3, BLOCK_SIZE_y_3> <<<n_blocks, n_threads>>> (1.0,
																			  reg,
																			  d_mat_dz1_t,
																			  d_mat_X,
																			  d_mat_W1,
																			  d_mat_dW1,
																			  W2_n_cols,
																			  y_n_rows,
																			  X_n_cols,
																			  false,
																			  false);

	/*
	 * Compute db1
	 */
	
	// Compute the block dimension
	n_threads.x = BLOCK_SIZE_REDUCTION;
	n_threads.y = 1;
	
	// Compute the grid size
	n_blocks.x = 1;
	n_blocks.y = y_n_rows;
	
	device_sum_col_block <BLOCK_SIZE_REDUCTION> <<<n_blocks, n_threads>>> (d_mat_diff, d_mat_db2, y_n_cols, y_n_rows);

	/*
	 * Compute db2
	 */
	
	// Compute the block dimension
	n_threads.x = BLOCK_SIZE_REDUCTION;
	n_threads.y = 1;
	
	// Compute the grid size
	n_blocks.x = 1;
	n_blocks.y = W2_n_cols;
	
	device_sum_col_block <BLOCK_SIZE_REDUCTION> <<<n_blocks, n_threads>>> (d_mat_dz1, d_mat_db1, y_n_rows, W2_n_cols);

	/*
	 * Copy data from the device memory to the host memory
	 */
	checkCudaErrors(hipMemcpy(mat_a2, d_mat_z2, X_n_rows*W2_n_rows*sizeof(double), hipMemcpyDeviceToHost));	
	checkCudaErrors(hipMemcpy(mat_dW1, d_mat_dW1, dW1_n_rows*dW1_n_cols*sizeof(double), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(mat_dW2, d_mat_dW2, dW2_n_rows*dW2_n_cols*sizeof(double), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(mat_db1, d_mat_db1, db1_n_cols*sizeof(double), hipMemcpyDeviceToHost));	
	checkCudaErrors(hipMemcpy(mat_db2, d_mat_db2, db2_n_cols*sizeof(double), hipMemcpyDeviceToHost));
	
	/*
	 * Free the device memory
	 */
	
	hipFree(d_mat_X);
	hipFree(d_mat_W1);
	hipFree(d_mat_b1);
	hipFree(d_mat_z1);
	hipFree(d_mat_W2);
	hipFree(d_mat_b2);
	hipFree(d_mat_z2);
	
	hipFree(d_mat_dz1);
	hipFree(d_mat_da1);
	hipFree(d_mat_dW1);
	hipFree(d_mat_dW2);
	hipFree(d_mat_db1);
	hipFree(d_mat_db2);
	
	hipFree(d_mat_y);
	hipFree(d_mat_diff);
	
	hipFree(d_mat_W1_t);
	hipFree(d_mat_W2_t);
	hipFree(d_mat_diff_t);
	hipFree(d_mat_dz1_t);
}
